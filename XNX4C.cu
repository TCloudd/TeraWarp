#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <malloc.h>
#include<time.h>
#include"newmat.h"
#include "ele.cuh"


//__global__ void get_bTWOKERNAL(const Matrixtran dev_q2, const Matrixtran dev_xnxn_K, Matrixtran  dev_A1)
//{
//	int row = blockIdx.y * blockDim.y + threadIdx.y;
//	int col = blockIdx.x * blockDim.x + threadIdx.x;
//
//	if (col >= dev_A1.width || row >= dev_A1.height)
//		return;
//	float sum = 0;
//	for (int i = 0; i < dev_q2.height; i++)
//	{
//		sum += dev_q2.elements[i*dev_q2.width + row] * dev_xnxn_K.elements[i*dev_xnxn_K.width + col];
//	}
//	dev_A1.elements[dev_A1.width*row + col] = sum;
//}

__global__ void get_c1(const Matrixtran dev_q2, const Matrixtran dev_A_t, Matrixtran  dev_C1)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col >= dev_C1.width || row >= dev_C1.height)
		return;
	float sum = 0;
	for (int i = 0; i < dev_q2.width; i++)
	{
		sum += dev_q2.elements[row*dev_q2.width + i] * dev_A_t.elements[i*dev_A_t.width + col];
	}
	dev_C1.elements[dev_C1.width*row + col] = sum;


}
__global__ void get_c2(Matrixtran dev_C1, const Matrixtran dev_q2, Matrixtran  dev_C2)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col >= dev_C2.width || row >= dev_C2.height)
		return;
	float sum = 0;
	for (int i = 0; i < dev_C1.width; i++)
	{
		sum += dev_C1.elements[row*dev_C1.width + i] * dev_q2.elements[col*dev_q2.width + i];
	}
	dev_C2.elements[dev_C2.width*row + col] = sum;
}
__global__ void get_c(Matrixtran dev_C2, const Matrixtran dev_Y, Matrixtran  dev_C)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col >= dev_C.width || row >= dev_C.height)
		return;
	float sum = 0;
	for (int i = 0; i < dev_C2.width; i++)
	{
		sum += dev_C2.elements[row*dev_C2.width + i] * dev_Y.elements[i*dev_Y.width + col];
	}
	dev_C.elements[dev_C.width*row + col] = sum;


}
//__global__ void get_ai(Matrixtran dev_A, Matrixtran  dev_A_i)
//{
//	int isx = blockIdx.x * blockDim.x + threadIdx.x;
//	int isy = blockIdx.y * blockDim.y + threadIdx.y;
//	double tmpIn;
//	double tmpInv;
//	//initialize E
//	if (isx == isy)
//		dev_A_i.elements[isy*dev_A_i.width+isx] = 1;
//	else
//		dev_A_i.elements[isy*dev_A_i.width + isx] = 0;
//
//	for (int i = 0; i < dev_A.height; i++)
//	{
//		if (i == isy && isx < (dev_A.width) && isy < dev_A.height)
//		{
//			//�����Խ����ϵ�Ԫ�أ���Ԫ��Ϊ1
//			tmpIn = dev_A.elements[isy*dev_A.width + isx];
//			dev_A.elements[i*dev_A.width + isx] /= tmpIn;
//			dev_A_i.elements[i*dev_A_i.width + isx] /= tmpIn;
//		}
//		__syncthreads();
//		if (i != isy && isx < 3 && isy < 3)
//		{
//			//����Ԫ�����е�Ԫ�ػ�Ϊ0 �����е�Ԫ��ͬʱ�仯
//			tmpInv = dev_A_i.elements[isy*dev_A_i.width + i];
//			dev_A.elements[isy*dev_A.width + isx] -= tmpInv * dev_A.elements[i*dev_A.width + isx];
//			dev_A_i.elements[isy*dev_A_i.width + isx] -= tmpInv * dev_A_i.elements[i*dev_A_i.width + isx];
//		}
//		__syncthreads();
//	}
//}
extern "C" bool gpu_xnxn(int ncpt, const Matrix &q2, const Matrix &A_t, Matrix &C)
{

	hipEvent_t   start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));
	printf("\t---------------------cuda process start---------------------\n");
	printf("\tncpt:%d\n", ncpt);
	//�����ڴ�
	Matrixtran dev_q2, dev_A_t, host_q2, host_A_t, host_Y, dev_Y, host_C1, dev_C1, host_C2, dev_C2;
	host_q2.width = ncpt - 4; host_q2.height = ncpt; size_t size_host_q2 = host_q2.width*host_q2.height*sizeof(float); host_q2.elements = (float*)malloc(size_host_q2);
	host_A_t.width = ncpt - 4; host_A_t.height = ncpt - 4; size_t size_host_A_t = host_A_t.width*host_A_t.height*sizeof(float); host_A_t.elements = (float*)malloc(size_host_A_t);
	dev_q2.width = ncpt - 4; dev_q2.height = ncpt; size_t size_dev_q2 = dev_q2.width*dev_q2.height*sizeof(float); hipMalloc((void**)&dev_q2.elements, size_dev_q2);
	dev_A_t.width = ncpt - 4; dev_A_t.height = ncpt - 4; size_t size_dev_A_t = dev_A_t.width*dev_A_t.height*sizeof(float); hipMalloc((void**)&dev_A_t.elements, size_dev_A_t);
	//host_Y.width = 4; host_Y.height = ncpt; size_t size_host_Y = host_Y.width*host_Y.height*sizeof(float); host_Y.elements = (float*)malloc(size_host_Y);
	//dev_Y.width = 4; dev_Y.height = ncpt; size_t size_dev_Y = dev_Y.width*dev_Y.height*sizeof(float); hipMalloc((void**)&dev_Y.elements, size_dev_Y);
	//host_C.width =4; host_C.height = ncpt; size_t size_host_C = host_C.width*host_C.height*sizeof(float); host_C.elements = (float*)malloc(size_host_C);
	//dev_C.width =4; dev_C.height = ncpt ; size_t size_dev_C = dev_C.width*dev_C.height*sizeof(float); hipMalloc((void**)&dev_C.elements, size_dev_C);
	host_C1.width = ncpt - 4; host_C1.height = ncpt; size_t size_host_C1 = host_C1.width*host_C1.height*sizeof(float); host_C1.elements = (float*)malloc(size_host_C1);
	dev_C1.width = ncpt - 4; dev_C1.height = ncpt; size_t size_dev_C1 = dev_C1.width*dev_C1.height*sizeof(float); hipMalloc((void**)&dev_C1.elements, size_dev_C1);
	host_C2.width = ncpt; host_C2.height = ncpt; size_t size_host_C2 = host_C2.width*host_C2.height*sizeof(float); host_C2.elements = (float*)malloc(size_host_C2);
	dev_C2.width = ncpt; dev_C2.height = ncpt; size_t size_dev_C2 = dev_C2.width*dev_C2.height*sizeof(float); hipMalloc((void**)&dev_C2.elements, size_dev_C2);

	//ת����������
	for (int i = 0; i < q2.nrows(); i++)
	{
		for (int j = 0; j < q2.ncols(); j++)
		{

			host_q2.elements[i*host_q2.width + j] = q2(i + 1, j + 1);

		}
	}

	for (int i = 0; i < A_t.nrows(); i++)
	{
		for (int j = 0; j < A_t.ncols(); j++)
		{
			host_A_t.elements[i*host_A_t.width + j] = A_t(i + 1, j + 1);

		}
	}

	//for (int i = 0; i < Y.nrows(); i++)
	//{
	//	for (int j = 0; j < Y.ncols(); j++)
	//	{
	//		host_Y.elements[i*host_Y.width + j] = Y(i + 1, j + 1);

	//	}
	//}

	//��cpu�����ڴ浽gpu
	HANDLE_ERROR(hipMemcpy(dev_q2.elements, host_q2.elements, size_host_q2, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_A_t.elements, host_A_t.elements, size_host_A_t, hipMemcpyHostToDevice));
	//HANDLE_ERROR(hipMemcpy(dev_Y.elements, host_Y.elements, size_host_Y, hipMemcpyHostToDevice));
	//HANDLE_ERROR(hipMemcpy(dev_C.elements, host_C.elements, size_host_C, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_C1.elements, host_C1.elements, size_host_C1, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_C2.elements, host_C2.elements, size_host_C2, hipMemcpyHostToDevice));

	dim3 devgrid((ncpt - 4 + threads_num - 1) / threads_num, (ncpt + threads_num - 1) / threads_num);
	dim3 devblock(threads_num, threads_num);
	dim3 devgrid_c((ncpt - 4 + threads_num - 1) / threads_num, 1);
	dim3 devblock_c(threads_num, 4);

	get_c1 << <devgrid, devblock >> >(dev_q2, dev_A_t, dev_C1);

	get_c2 << <devgrid, devblock >> >(dev_C1, dev_q2, dev_C2);
	//get_c << <devgrid_c, devblock_c >> >(dev_C2, dev_Y, dev_C);
	//��gpu�����ڴ浽cpu
	//HANDLE_ERROR(hipMemcpy(host_A1.elements, dev_A1.elements, dev_A1.width* dev_A1.height * sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(host_C2.elements, dev_C2.elements, size_dev_C2, hipMemcpyDeviceToHost));
	//printf("\n");
	//for (int j = 0; j < size_dev_C; j++)
	//{
	//	
	//	printf("%.3f\t", host_C.elements[j]);
	//	if (((j + 1) % host_C.width) == 0){ printf("\n��%d��", (j + 1) / host_C.width); }
	//
	//}


	//ת������
	for (int i = 0; i < C.nrows(); i++)
	{
		for (int j = 0; j < C.ncols(); j++)
		{
			C(i + 1, j + 1) = host_C2.elements[i*host_C2.width + j];

		}
	}
	//for (long long row = 1; row <= C.nrows(); row++)
	//{
	//	printf("\n��%d��", row);
	//	for (long long col = 1; col <= C.ncols(); col++)
	//		printf("%.3f\t", C(row, col));
	//	printf("\n");
	//}
	//��ʱ����
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("\tTime to generate: %3.1f ms\n", elapsedTime);
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	printf("\t---------------------cuda process end---------------------\n");
	//�ͷ��ڴ�
	hipFree(&dev_q2);
	hipFree(&dev_A_t);
	hipFree(&dev_C1);
	//hipFree(&dev_Y);
	hipFree(&dev_C2);
	HANDLE_ERROR(hipDeviceReset());
	return true;
}
