#include "hip/hip_runtime.h"
﻿#include "ele.cuh"
#include "hip/hip_runtime_api.h"
#include <malloc.h>
#include <time.h>
#include"newmat.h"
#include <math.h>
//#include "q_warp_affine_tps.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipsolver.h"
#include <assert.h>
#include <stdlib.h>
//#include <QtGui>

extern "C" Matrix matrixMultiply(const int m, const int n, const int k, Matrix &A, Matrix &B);

#define number 5000   //Ua_size in TPS
#define EPS 0.0001

__host__ __device__ void MUL_aff(float *A, float *B, float *C)//A*B=C
{
	for (int i = 0; i < 4; i++)
	{


		float sum = 0;
		for (int k = 0; k < 4; k++)
		{
			sum += A[i * 4 + k] * B[k];
		}
		C[i] = sum;

	}
}


extern "C" Matrix matrixMultiply(const int m, const int n, const int k, Matrix &A, Matrix &B)
{
	//A*B=C
	//m:A.row;n:B.col;k:A.col
	Matrix C(m, n);
	hipError_t cudaStat;
	hipblasStatus_t stat;
	float *H_A, *H_B, *H_C;

	float *D_A, *D_B, *D_C;

	H_A = (float*)malloc(m * k * sizeof(float));
	H_B = (float*)malloc(k * n * sizeof(float));
	H_C = (float*)malloc(m * n * sizeof(float));

	cudaStat = hipMalloc((void**)&D_A, m * k * sizeof(float));
	cudaStat = hipMalloc((void**)&D_B, k * n * sizeof(float));
	cudaStat = hipMalloc((void**)&D_C, m * n * sizeof(float));

	/*	cudaStat = hipMalloc((void**)&D_A, r_size * r_size * sizeof(float));
	cudaStat = hipMalloc((void**)&D_B, r_size * r_size * sizeof(float));
	cudaStat = hipMalloc((void**)&D_C, r_size * r_size * sizeof(float));*/
	printf("cudaStat %d\n", cudaStat);
	for (int i = 0; i < A.nrows(); i++)
	{
		for (int j = 0; j < A.ncols(); j++)
		{

			H_A[i * A.ncols() + j] = A(i + 1, j + 1);
			//if (H_A[i * A.ncols() + j] > EPS)printf("%f", H_A[i * A.ncols() + j]);
		}

	}

	for (int i = 0; i < B.nrows(); i++)
	{
		for (int j = 0; j < B.ncols(); j++)
		{

			H_B[i * B.ncols() + j] = B(i + 1, j + 1);
			//if (H_B[i * B.ncols() + j] > EPS)printf("%.2f\n", H_B[i * B.ncols() + j]);
		}

	}

	HANDLE_ERROR(hipMemcpy(D_A, H_A, m * k * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_B, H_B, k * n * sizeof(float), hipMemcpyHostToDevice));

	//	stat = hipblasSetMatrix(r_size, r_size, sizeof(*H_A), H_A, r_size, D_A, r_size);
	//	stat = hipblasSetMatrix(r_size, r_size, sizeof(*H_B), H_B, r_size, D_B, r_size);
	//	stat = hipblasSetMatrix(r_size, r_size, sizeof(*H_C), H_C, r_size, D_C, r_size);

	const float alpha = 1.0f;
	const float beta = 0.0f;

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, D_B, n, D_A, k, &beta, D_C, n);

	hipblasDestroy(handle);

	printf("cublas %d\n", stat);
	HANDLE_ERROR(hipMemcpy(H_C, D_C, m * n * sizeof(float), hipMemcpyDeviceToHost));


	for (int i = 0; i < C.nrows(); i++)
	{
		for (int j = 0; j < C.ncols(); j++)
		{

			C(i + 1, j + 1) = H_C[i * C.ncols() + j];
			//if (C(i + 1, j + 1) > EPS)printf("%f\n", C(i + 1, j + 1));
		}

	}

	/*	for (int i = 0; i < C.nrows(); i++)
	{
	for (int j = 0; j < C.ncols(); j++)
	{

	C(i + 1, j + 1) = H_C[j * C.ncols() + i];

	}

	}*/

	free(H_A);
	free(H_B);
	free(H_C);
	hipFree(D_A);
	hipFree(D_B);
	hipFree(D_C);
	HANDLE_ERROR(hipDeviceReset());
	return C;
}


__global__ void get_Displacement_affine_new_Z(const int k, const long long gsz1, const long long gsz0, float *D_x4x4_affinematrix, const int D_sz_img_sub0,
	const int D_sz_img_sub1, const int D_sz_img_sub2, const int D_sz_img_sub3, unsigned char *D_p_img_sub_4d, unsigned char *D_p_img_sub2tar_4d, const int D_sz_img_A_sub0,
	const int D_sz_img_A_sub1, const int D_sz_img_A_sub2, const long long start_block_x, const long long start_block_y, const long long start_block_z,
	const long long x_read_offset, const long long y_read_offset, const long long z_read_offset, const long long gs_ori2, const long long gs_ori1, const long long gs_ori0)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= gsz1 || col >= gsz0)return;


	float x_pt_sub2tar_homo[4];
	float x_pt_sub_homo[4];


	x_pt_sub2tar_homo[0] = col + start_block_x;
	x_pt_sub2tar_homo[1] = row + start_block_y;
	x_pt_sub2tar_homo[2] = k + start_block_z;
	x_pt_sub2tar_homo[3] = 1.0;

	MUL_aff(D_x4x4_affinematrix, x_pt_sub2tar_homo, x_pt_sub_homo);

	double cur_pos[3];//x,y,z
	double cur_pos_1[3];//x,y,z
	cur_pos[0] = x_pt_sub_homo[0] - x_read_offset;
	cur_pos[1] = x_pt_sub_homo[1] - y_read_offset;
	cur_pos[2] = x_pt_sub_homo[2] - z_read_offset;

	cur_pos_1[0] = x_pt_sub_homo[0];
	cur_pos_1[1] = x_pt_sub_homo[1];
	cur_pos_1[2] = x_pt_sub_homo[2];

	if (cur_pos_1[0]<0 || cur_pos_1[0]>gs_ori0 - 1 ||
		cur_pos_1[1]<0 || cur_pos_1[1]>gs_ori1 - 1 ||
		cur_pos_1[2]<0 || cur_pos_1[2]>gs_ori2 - 1 ||
		cur_pos[0]<0 || cur_pos[0]>D_sz_img_A_sub0 - 1 ||
		cur_pos[1]<0 || cur_pos[1]>D_sz_img_A_sub1 - 1 ||
		cur_pos[2]<0 || cur_pos[2]>D_sz_img_A_sub2 - 1)
	{
		for (long long c = 0; c < D_sz_img_sub3; c++)
		{
			D_p_img_sub2tar_4d[c * D_sz_img_sub2 * D_sz_img_sub1 * D_sz_img_sub0 + k * D_sz_img_sub1 * D_sz_img_sub0 + row * D_sz_img_sub0 + col] = 0.0;

		}

	}
	else{


		long long x_s, x_b, y_s, y_b, z_s, z_b;


		x_s = floor(cur_pos[0]);		x_b = ceil(cur_pos[0]);
		y_s = floor(cur_pos[1]);		y_b = ceil(cur_pos[1]);
		z_s = floor(cur_pos[2]);		z_b = ceil(cur_pos[2]);


		//compute weight for left and right, top and bottom -- 4 neighbor pixel's weight in a slice
		double l_w, r_w, t_w, b_w;
		l_w = 1.0 - (cur_pos[0] - x_s);	r_w = 1.0 - l_w;
		t_w = 1.0 - (cur_pos[1] - y_s);	b_w = 1.0 - t_w;
		//compute weight for higher slice and lower slice
		double u_w, d_w;
		u_w = 1.0 - (cur_pos[2] - z_s);	d_w = 1.0 - u_w;


		float a = t_w*(l_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_b]);

		float b = t_w*(l_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_b]);

		long long d = k * D_sz_img_sub1 * D_sz_img_sub0 + row * D_sz_img_sub0 + col;

		D_p_img_sub2tar_4d[d] = u_w*a + d_w*b;



	}

}


__global__ void get_Displacement_affine_new_Y(const int k, const long long gsz1, const long long gsz0, float *D_x4x4_affinematrix, const int D_sz_img_sub0,
	const int D_sz_img_sub1, const int D_sz_img_sub2, const int D_sz_img_sub3, unsigned char *D_p_img_sub_4d, unsigned char *D_p_img_sub2tar_4d, const int D_sz_img_A_sub0,
	const int D_sz_img_A_sub1, const int D_sz_img_A_sub2, const long long start_block_x, const long long start_block_y, const long long start_block_z,
	const long long x_read_offset, const long long y_read_offset, const long long z_read_offset, const long long gs_ori2, const long long gs_ori1, const long long gs_ori0)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= gsz1 || col >= gsz0)return;


	float x_pt_sub2tar_homo[4];
	float x_pt_sub_homo[4];


	x_pt_sub2tar_homo[0] = col + start_block_x;
	x_pt_sub2tar_homo[1] = k + start_block_y;
	x_pt_sub2tar_homo[2] = row + start_block_z;
	x_pt_sub2tar_homo[3] = 1.0;

	MUL_aff(D_x4x4_affinematrix, x_pt_sub2tar_homo, x_pt_sub_homo);

	double cur_pos[3];//x,y,z
	double cur_pos_1[3];//x,y,z
	cur_pos[0] = x_pt_sub_homo[0] - x_read_offset;
	cur_pos[1] = x_pt_sub_homo[1] - y_read_offset;
	cur_pos[2] = x_pt_sub_homo[2] - z_read_offset;

	cur_pos_1[0] = x_pt_sub_homo[0];
	cur_pos_1[1] = x_pt_sub_homo[1];
	cur_pos_1[2] = x_pt_sub_homo[2];

	if (cur_pos_1[0]<0 || cur_pos_1[0]>gs_ori0 - 1 ||
		cur_pos_1[1]<0 || cur_pos_1[1]>gs_ori1 - 1 ||
		cur_pos_1[2]<0 || cur_pos_1[2]>gs_ori2 - 1 ||
		cur_pos[0]<0 || cur_pos[0]>D_sz_img_A_sub0 - 1 ||
		cur_pos[1]<0 || cur_pos[1]>D_sz_img_A_sub1 - 1 ||
		cur_pos[2]<0 || cur_pos[2]>D_sz_img_A_sub2 - 1)
	{
		for (long long c = 0; c < D_sz_img_sub3; c++)
		{
			D_p_img_sub2tar_4d[c * D_sz_img_sub2 * D_sz_img_sub1 * D_sz_img_sub0 + row * D_sz_img_sub1 * D_sz_img_sub0 + k * D_sz_img_sub0 + col] = 0.0;

		}

	}
	else{


		long long x_s, x_b, y_s, y_b, z_s, z_b;


		x_s = floor(cur_pos[0]);		x_b = ceil(cur_pos[0]);
		y_s = floor(cur_pos[1]);		y_b = ceil(cur_pos[1]);
		z_s = floor(cur_pos[2]);		z_b = ceil(cur_pos[2]);


		//compute weight for left and right, top and bottom -- 4 neighbor pixel's weight in a slice
		double l_w, r_w, t_w, b_w;
		l_w = 1.0 - (cur_pos[0] - x_s);	r_w = 1.0 - l_w;
		t_w = 1.0 - (cur_pos[1] - y_s);	b_w = 1.0 - t_w;
		//compute weight for higher slice and lower slice
		double u_w, d_w;
		u_w = 1.0 - (cur_pos[2] - z_s);	d_w = 1.0 - u_w;


		float a = t_w*(l_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_b]);

		float b = t_w*(l_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_b]);

		long long d = row * D_sz_img_sub1 * D_sz_img_sub0 + k * D_sz_img_sub0 + col;

		D_p_img_sub2tar_4d[d] = u_w*a + d_w*b;



	}

}

__global__ void get_Displacement_affine_new_X(const int k, const long long gsz1, const long long gsz0, float *D_x4x4_affinematrix, const int D_sz_img_sub0,
	const int D_sz_img_sub1, const int D_sz_img_sub2, const int D_sz_img_sub3, unsigned char *D_p_img_sub_4d, unsigned char *D_p_img_sub2tar_4d, const int D_sz_img_A_sub0,
	const int D_sz_img_A_sub1, const int D_sz_img_A_sub2, const long long start_block_x, const long long start_block_y, const long long start_block_z,
	const long long x_read_offset, const long long y_read_offset, const long long z_read_offset, const long long gs_ori2, const long long gs_ori1, const long long gs_ori0)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= gsz1 || col >= gsz0)return;


	float x_pt_sub2tar_homo[4];
	float x_pt_sub_homo[4];


	x_pt_sub2tar_homo[0] = k + start_block_x;
	x_pt_sub2tar_homo[1] = col + start_block_y;
	x_pt_sub2tar_homo[2] = row + start_block_z;
	x_pt_sub2tar_homo[3] = 1.0;

	MUL_aff(D_x4x4_affinematrix, x_pt_sub2tar_homo, x_pt_sub_homo);

	double cur_pos[3];//x,y,z
	double cur_pos_1[3];//x,y,z
	cur_pos[0] = x_pt_sub_homo[0] - x_read_offset;
	cur_pos[1] = x_pt_sub_homo[1] - y_read_offset;
	cur_pos[2] = x_pt_sub_homo[2] - z_read_offset;

	cur_pos_1[0] = x_pt_sub_homo[0];
	cur_pos_1[1] = x_pt_sub_homo[1];
	cur_pos_1[2] = x_pt_sub_homo[2];

	if (cur_pos_1[0]<0 || cur_pos_1[0]>gs_ori0 - 1 ||
		cur_pos_1[1]<0 || cur_pos_1[1]>gs_ori1 - 1 ||
		cur_pos_1[2]<0 || cur_pos_1[2]>gs_ori2 - 1 ||
		cur_pos[0]<0 || cur_pos[0]>D_sz_img_A_sub0 - 1 ||
		cur_pos[1]<0 || cur_pos[1]>D_sz_img_A_sub1 - 1 ||
		cur_pos[2]<0 || cur_pos[2]>D_sz_img_A_sub2 - 1)
	{
		for (long long c = 0; c < D_sz_img_sub3; c++)
		{
			D_p_img_sub2tar_4d[c * D_sz_img_sub2 * D_sz_img_sub1 * D_sz_img_sub0 + row * D_sz_img_sub1 * D_sz_img_sub0 + col * D_sz_img_sub0 + k] = 0.0;

		}

	}
	else{


		long long x_s, x_b, y_s, y_b, z_s, z_b;


		x_s = floor(cur_pos[0]);		x_b = ceil(cur_pos[0]);
		y_s = floor(cur_pos[1]);		y_b = ceil(cur_pos[1]);
		z_s = floor(cur_pos[2]);		z_b = ceil(cur_pos[2]);


		//compute weight for left and right, top and bottom -- 4 neighbor pixel's weight in a slice
		double l_w, r_w, t_w, b_w;
		l_w = 1.0 - (cur_pos[0] - x_s);	r_w = 1.0 - l_w;
		t_w = 1.0 - (cur_pos[1] - y_s);	b_w = 1.0 - t_w;
		//compute weight for higher slice and lower slice
		double u_w, d_w;
		u_w = 1.0 - (cur_pos[2] - z_s);	d_w = 1.0 - u_w;


		float a = t_w*(l_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_b]);

		float b = t_w*(l_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_b]);

		long long d = row * D_sz_img_sub1 * D_sz_img_sub0 + col * D_sz_img_sub0 + k;

		D_p_img_sub2tar_4d[d] = u_w*a + d_w*b;



	}

}



__global__ void get_Displacement_affine(const int k, const long long gsz1, const long long gsz0, float *D_x4x4_affinematrix, const int D_sz_img_sub0,
	const int D_sz_img_sub1, const int D_sz_img_sub2, const int D_sz_img_sub3, float *D_p_img_sub_4d, float *D_p_img_sub2tar_4d, const int D_sz_img_A_sub0,
	const int D_sz_img_A_sub1, const int D_sz_img_A_sub2)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= gsz1 || col >= gsz0)return;

	float x_pt_sub2tar_homo[4];
	float x_pt_sub_homo[4];


	x_pt_sub2tar_homo[0] = col;
	x_pt_sub2tar_homo[1] = row;
	x_pt_sub2tar_homo[2] = k;
	x_pt_sub2tar_homo[3] = 1.0;

	MUL_aff(D_x4x4_affinematrix, x_pt_sub2tar_homo, x_pt_sub_homo);

	double cur_pos[3];//x,y,z
	cur_pos[0] = x_pt_sub_homo[0];
	cur_pos[1] = x_pt_sub_homo[1];
	cur_pos[2] = x_pt_sub_homo[2];

	if (cur_pos[0]<0 || cur_pos[0]>D_sz_img_A_sub0 - 1 ||
		cur_pos[1]<0 || cur_pos[1]>D_sz_img_A_sub1 - 1 ||
		cur_pos[2]<0 || cur_pos[2]>D_sz_img_A_sub2 - 1)
	{
		for (long long c = 0; c < D_sz_img_sub3; c++)
		{
			D_p_img_sub2tar_4d[c * D_sz_img_sub2 * D_sz_img_sub1 * D_sz_img_sub0 + k * D_sz_img_sub1 * D_sz_img_sub0 + row * D_sz_img_sub0 + col] = 0.0;

		}

	}
	else{


		long long x_s, x_b, y_s, y_b, z_s, z_b;


		x_s = floor(cur_pos[0]);		x_b = ceil(cur_pos[0]);
		y_s = floor(cur_pos[1]);		y_b = ceil(cur_pos[1]);
		z_s = floor(cur_pos[2]);		z_b = ceil(cur_pos[2]);


		//compute weight for left and right, top and bottom -- 4 neighbor pixel's weight in a slice
		double l_w, r_w, t_w, b_w;
		l_w = 1.0 - (cur_pos[0] - x_s);	r_w = 1.0 - l_w;
		t_w = 1.0 - (cur_pos[1] - y_s);	b_w = 1.0 - t_w;
		//compute weight for higher slice and lower slice
		double u_w, d_w;
		u_w = 1.0 - (cur_pos[2] - z_s);	d_w = 1.0 - u_w;


		float a = t_w*(l_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_b]);

		float b = t_w*(l_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_s*D_sz_img_A_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b*D_sz_img_A_sub1 * D_sz_img_A_sub0 + y_b*D_sz_img_A_sub0 + x_b]);

		int d = k * D_sz_img_sub1 * D_sz_img_sub0 + row * D_sz_img_sub0 + col;

		D_p_img_sub2tar_4d[d] = u_w*a + d_w*b;



	}

}

extern "C" bool gpu_interpolation_new(const int mode, const long long gsz2, const long long gsz1, const long long gsz0, const Matrix &x4x4_affinematrix, unsigned char ****&p_img_sub_4d,
	unsigned char ****&p_img_sub2tar_4d, const long long *sz_img_sub, const long long gsA2, const long long gsA1, const long long gsA0, const long long start_block_x, const long long start_block_y,
	const long long start_block_z, const long long x_read_offset, const long long y_read_offset, const long long z_read_offset, const long long gs_ori2, const long long gs_ori1, const long long gs_ori0,
	const unsigned char *p_img_sub, unsigned char *p_img_affine)
{


	float *H_x4x4_affinematrix, *H_sz_img_sub;

	float *D_x4x4_affinematrix, *D_H_sz_img_sub;

	unsigned char *D_p_img_sub_4d, *D_p_img_sub2tar_4d;

	H_x4x4_affinematrix = (float*)malloc(x4x4_affinematrix.nrows() * x4x4_affinematrix.ncols() * sizeof(float));



	hipMalloc((void**)&D_x4x4_affinematrix, x4x4_affinematrix.nrows() * x4x4_affinematrix.ncols() * sizeof(float));
	hipMalloc((void**)&D_p_img_sub_4d, gsA0 * gsA1 * gsA2 * sz_img_sub[3] * sizeof(unsigned char));
	hipMalloc((void**)&D_p_img_sub2tar_4d, sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(unsigned char));


	int aa = sz_img_sub[0]; int bb = sz_img_sub[1]; int cc = sz_img_sub[2]; int dd = sz_img_sub[3];


	for (int i = 0; i < x4x4_affinematrix.nrows(); i++)
	{
		for (int j = 0; j < x4x4_affinematrix.ncols(); j++)
		{

			H_x4x4_affinematrix[i * x4x4_affinematrix.ncols() + j] = x4x4_affinematrix(i + 1, j + 1);

		}

	}

	HANDLE_ERROR(hipMemcpy(D_x4x4_affinematrix, H_x4x4_affinematrix, x4x4_affinematrix.nrows() * x4x4_affinematrix.ncols() * sizeof(float), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpy(D_p_img_sub_4d, p_img_sub, gsA0 * gsA1 * gsA2 * sz_img_sub[3] * sizeof(unsigned char), hipMemcpyHostToDevice));
	/*
	*mode:0; z:gsz2; y:gsz1; x:gsz0; z=k;y=row;x=col;
	*mode:1; y:gsz2; z:gsz1; x:gsz0; y=k;z=row;x=col;
	*mode:2; x:gsz2; z:gsz1; y:gsz0; x=k;z=row;y=col;
	*/
	switch (mode)
	{
	case 0:
		for (long long k = 0; k < gsz2; k++)
		{
			dim3 grid((gsz0 + threads_num - 1) / threads_num, (gsz1 + threads_num - 1) / threads_num);
			dim3 block(threads_num, threads_num);
			//get_Displacement_affine_new << <grid, block >> >(k, bb_real, aa_real, D_x4x4_affinematrix, aa, bb, cc, dd, D_p_img_sub_4d, D_p_img_sub2tar_4d, gsA0, gsA1, gsA2, start_block_x, start_block_y, start_block_z, x_read_offset, y_read_offset, z_read_offset, gs_ori2, gs_ori1, gs_ori0, aa_real, bb_real, cc_real);
			get_Displacement_affine_new_Z << <grid, block >> >(k, gsz1, gsz0, D_x4x4_affinematrix, aa, bb, cc, dd, D_p_img_sub_4d, D_p_img_sub2tar_4d, gsA0, gsA1, gsA2, start_block_x, start_block_y, start_block_z, x_read_offset, y_read_offset, z_read_offset, gs_ori2, gs_ori1, gs_ori0);
			hipDeviceSynchronize();
		}
		break;
	case 1:
		for (long long k = 0; k < gsz2; k++)
		{
			dim3 grid((gsz0 + threads_num - 1) / threads_num, (gsz1 + threads_num - 1) / threads_num);
			dim3 block(threads_num, threads_num);
			//get_Displacement_affine_new << <grid, block >> >(k, bb_real, aa_real, D_x4x4_affinematrix, aa, bb, cc, dd, D_p_img_sub_4d, D_p_img_sub2tar_4d, gsA0, gsA1, gsA2, start_block_x, start_block_y, start_block_z, x_read_offset, y_read_offset, z_read_offset, gs_ori2, gs_ori1, gs_ori0, aa_real, bb_real, cc_real);
			get_Displacement_affine_new_Y << <grid, block >> >(k, gsz1, gsz0, D_x4x4_affinematrix, aa, bb, cc, dd, D_p_img_sub_4d, D_p_img_sub2tar_4d, gsA0, gsA1, gsA2, start_block_x, start_block_y, start_block_z, x_read_offset, y_read_offset, z_read_offset, gs_ori2, gs_ori1, gs_ori0);
			hipDeviceSynchronize();
		}
		break;
	case 2:
		for (long long k = 0; k < gsz2; k++)
		{
			dim3 grid((gsz0 + threads_num - 1) / threads_num, (gsz1 + threads_num - 1) / threads_num);
			dim3 block(threads_num, threads_num);
			//get_Displacement_affine_new << <grid, block >> >(k, bb_real, aa_real, D_x4x4_affinematrix, aa, bb, cc, dd, D_p_img_sub_4d, D_p_img_sub2tar_4d, gsA0, gsA1, gsA2, start_block_x, start_block_y, start_block_z, x_read_offset, y_read_offset, z_read_offset, gs_ori2, gs_ori1, gs_ori0, aa_real, bb_real, cc_real);
			get_Displacement_affine_new_X << <grid, block >> >(k, gsz1, gsz0, D_x4x4_affinematrix, aa, bb, cc, dd, D_p_img_sub_4d, D_p_img_sub2tar_4d, gsA0, gsA1, gsA2, start_block_x, start_block_y, start_block_z, x_read_offset, y_read_offset, z_read_offset, gs_ori2, gs_ori1, gs_ori0);
			hipDeviceSynchronize();
		}
		break;
	default:
		printf("Error:calculate mode is wrong!\n"); return false;

	}

	HANDLE_ERROR(hipMemcpy(p_img_affine, D_p_img_sub2tar_4d, sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(unsigned char), hipMemcpyDeviceToHost));

	free(H_x4x4_affinematrix);
	hipFree(D_x4x4_affinematrix); hipFree(D_p_img_sub_4d); hipFree(D_p_img_sub2tar_4d);
	HANDLE_ERROR(hipDeviceReset());
	return true;
}


extern "C" bool gpu_interpolation_affine(const long long gsz2, const long long gsz1, const long long gsz0, const Matrix &x4x4_affinematrix, unsigned char ****&p_img_sub_4d, unsigned char ****&p_img_sub2tar_4d, const long long *sz_img_sub,
	const long long gsA2, const long long gsA1, const long long gsA0)
{


	float *H_x4x4_affinematrix, *H_p_img_sub_4d, *H_sz_img_sub, *H_p_img_sub2tar_4d;

	float *D_x4x4_affinematrix, *D_p_img_sub_4d, *D_H_sz_img_sub, *D_p_img_sub2tar_4d;


	H_x4x4_affinematrix = (float*)malloc(x4x4_affinematrix.nrows() * x4x4_affinematrix.ncols() * sizeof(float));
	H_p_img_sub_4d = (float*)malloc(2 * gsA0 * gsA1 * gsA2 * sz_img_sub[3] * sizeof(float));
	H_p_img_sub2tar_4d = (float*)malloc(2 * sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float));


	hipMalloc((void**)&D_x4x4_affinematrix, x4x4_affinematrix.nrows() * x4x4_affinematrix.ncols() * sizeof(float));
	hipMalloc((void**)&D_p_img_sub_4d, 2 * gsA0 * gsA1 * gsA2 * sz_img_sub[3] * sizeof(float));
	hipMalloc((void**)&D_p_img_sub2tar_4d, 2 * sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float));


	int aa = sz_img_sub[0]; int bb = sz_img_sub[1]; int cc = sz_img_sub[2]; int dd = sz_img_sub[3];

	for (int i = 0; i < x4x4_affinematrix.nrows(); i++)
	{
		for (int j = 0; j < x4x4_affinematrix.ncols(); j++)
		{

			H_x4x4_affinematrix[i * x4x4_affinematrix.ncols() + j] = x4x4_affinematrix(i + 1, j + 1);

		}

	}

	for (long long a = 0; a <gsA2; a++)
	{
		for (long long b = 0; b < gsA1; b++)
		{
			for (long long c = 0; c < gsA0; c++)
			{
				H_p_img_sub_4d[a*gsA1 * gsA0 + b*gsA0 + c] = p_img_sub_4d[0][a][b][c];

			}
		}
	}//ע�⣺H_p_img_sub_4d��H_p_img_sub2tar_4d��ÿ��ά�ȴ�С��ͬ




	hipMemcpy(D_x4x4_affinematrix, H_x4x4_affinematrix, x4x4_affinematrix.nrows() * x4x4_affinematrix.ncols() * sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(D_p_img_sub_4d, H_p_img_sub_4d, 2 * gsA0 * gsA1 * gsA2 * sz_img_sub[3] * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(D_p_img_sub2tar_4d, H_p_img_sub2tar_4d, 2 * sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float), hipMemcpyHostToDevice);





	for (long long k = 0; k < gsz2; k++)
	{
		dim3 grid((gsz0 + threads_num - 1) / threads_num, (gsz1 + threads_num - 1) / threads_num);
		dim3 block(threads_num, threads_num);
		get_Displacement_affine << <grid, block >> >(k, gsz1, gsz0, D_x4x4_affinematrix, aa, bb, cc, dd, D_p_img_sub_4d, D_p_img_sub2tar_4d, gsA0, gsA1, gsA2);
		hipDeviceSynchronize();
	}


	hipMemcpy(H_p_img_sub2tar_4d, D_p_img_sub2tar_4d, 2 * sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float), hipMemcpyDeviceToHost);

	for (long long a = 0; a < sz_img_sub[2]; a++)
	{
		for (long long b = 0; b < sz_img_sub[1]; b++)
		{
			for (long long c = 0; c < sz_img_sub[0]; c++)
			{
				p_img_sub2tar_4d[0][a][b][c] = H_p_img_sub2tar_4d[a*sz_img_sub[1] * sz_img_sub[0] + b*sz_img_sub[0] + c];
			}
		}
	}

	hipFree(D_x4x4_affinematrix); hipFree(D_p_img_sub_4d); hipFree(D_p_img_sub2tar_4d);
	return true;
}