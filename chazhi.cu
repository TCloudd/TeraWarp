#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <malloc.h>
#include <iostream>
#include <chrono>
#include <ctime>
#include <time.h>
#include <stdlib.h>
#include"newmat.h"
#include "ele.cuh"

#if defined(_MSC_VER) && (_WIN64)

#define V3DLONG long long

#else

#define V3DLONG long

#endif



typedef unsigned char UINT8_JBA;
typedef float MYFLOAT_JBA;

class DisplaceFieldF3D
{
public:
	UINT8_JBA b_transform;

	MYFLOAT_JBA sx, sy, sz; //shift of x,y,z
	DisplaceFieldF3D() { sx = sy = sz = 0; b_transform = 0; }
	DisplaceFieldF3D(double vv) { sx = sy = sz = vv; b_transform = 0; }
	void scale(double dfactor) { sx *= dfactor; sy *= dfactor; sz *= dfactor; }
	void resetToDefault() //070517
	{
		sx = 0; sy = 0; sz = 0;
		b_transform = 0;
	}
	bool copy(DisplaceFieldF3D *wp)
	{
		if (!wp) return false;
		sx = wp->sx;  sy = wp->sy;  sz = wp->sz;
		b_transform = wp->b_transform;
		return true;
	}
	bool copy(DisplaceFieldF3D &wp)
	{
		sx = wp.sx;  sy = wp.sy;  sz = wp.sz;
		b_transform = wp.b_transform;
		return true;
	}

};

__device__ int myceil(double x)//����ȡ��
{
	if (x<0)

		return (float)((int)x);

	else

	{

		if (x == 0)

			return (float)((int)x) + 1;

		else

			return (float)((int)x) + 1;

	}
}
__device__ int myfloor(double x)//����ȡ��
{
	if (x<0)

		return (float)((int)x) - 1;

	else

	{

		if (x == 0)

			return (float)((int)x);

		else

			return (float)((int)x);

	}
}
__device__ void MUL(float *A, float *B, float *C)
{
	for (int i = 0; i < 64; i++)
	{

		for (int j = 0; j < 3; j++)
		{
			float sum = 0;
			for (int k = 0; k < 64; k++)
			{
				sum += A[i * 64 + k] * B[k * 3 + j];
			}
			C[i * 3 + j] = sum;
		}
	}
}


__global__ void get_Displacement_brightness(const V3DLONG sz_gridwnd, const int k, const V3DLONG gsz1, const V3DLONG gsz0, float * D_pppSubDF_x, float * D_pppSubDF_y, float * D_pppSubDF_z,
	float *D_x_bsplinebasis, const int szBlock_x, const int szBlock_y, const int szBlock_z, const int D_sz_img_sub0, const int D_sz_img_sub1, const int D_sz_img_sub2, const int D_sz_img_sub3, unsigned char *D_p_img_sub_4d, unsigned char *D_p_img_warp_4d,
	const int D_sz_img_read_sub0, const int D_sz_img_read_sub1, const int D_sz_img_read_sub2, const long long start_block_x, const long long start_block_y, const long long start_block_z, const long long x_read_offset, const long long y_read_offset, const long long z_read_offset, const long long gs_ori2, const long long gs_ori1,
	const long long gs_ori0)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;
//	printf("grid[%d %d]\n", row, col);
	if (row >= gsz1 - 1 - 2 || col >= gsz0 - 1 - 2) return;

	float x1D_gridblock[192];
	int ind = 1;
	for (int a = k; a<k + 4; a++)
		for (int c = col; c<col + 4; c++)
			for (int b = row; b<row + 4; b++)
			{
				x1D_gridblock[(ind - 1) * 3] = D_pppSubDF_x[a*gsz1*gsz0 + b*gsz0 + c];
				x1D_gridblock[(ind - 1) * 3 + 1] = D_pppSubDF_y[a*gsz1*gsz0 + b*gsz0 + c];
				x1D_gridblock[(ind - 1) * 3 + 2] = D_pppSubDF_z[a*gsz1*gsz0 + b*gsz0 + c];
				ind++;
			}


	float  x1D_gridblock_int[4 * 4 * 4 * 3];


	MUL(D_x_bsplinebasis, x1D_gridblock, x1D_gridblock_int);

	int idx = 1;
	float D_pppDFBlock_x1[4 * 4 * 4]; float D_pppDFBlock_y1[4 * 4 * 4]; float D_pppDFBlock_z1[4 * 4 * 4];
	for (long zz = 0; zz<sz_gridwnd; zz++)
		for (long xx = 0; xx<sz_gridwnd; xx++)
			for (long yy = 0; yy<sz_gridwnd; yy++)
			{
				D_pppDFBlock_x1[zz * 16 + yy * 4 + xx] = x1D_gridblock_int[(idx - 1) * 3];
				D_pppDFBlock_y1[zz * 16 + yy * 4 + xx] = x1D_gridblock_int[(idx - 1) * 3 + 1];
				D_pppDFBlock_z1[zz * 16 + yy * 4 + xx] = x1D_gridblock_int[(idx - 1) * 3 + 2];
				idx++;
			}
	long long start_x, start_y, start_z;
	start_x = col*szBlock_x;
	start_y = row*szBlock_y;
	start_z = k*szBlock_z;

	for (int z = 0; z < szBlock_z; z++)
		for (int y = 0; y < szBlock_y; y++)
			for (int x = 0; x < szBlock_x; x++)
			{

				long long pos_warp[3];
				pos_warp[0] = start_x + x;
				pos_warp[1] = start_y + y;
				pos_warp[2] = start_z + z;


				if (pos_warp[0] >= D_sz_img_sub0 || pos_warp[1] >= D_sz_img_sub1 || pos_warp[2] >= D_sz_img_sub2)
					continue;

				double pos_sub[3], pos_sub_l[3];
				pos_sub[0] = pos_warp[0] + D_pppDFBlock_x1[z * 16 + y * 4 + x] + start_block_x - x_read_offset;
				pos_sub[1] = pos_warp[1] + D_pppDFBlock_y1[z * 16 + y * 4 + x] + start_block_y - y_read_offset;
				pos_sub[2] = pos_warp[2] + D_pppDFBlock_z1[z * 16 + y * 4 + x] + start_block_z - z_read_offset;

				pos_sub_l[0] = pos_warp[0] + D_pppDFBlock_x1[z * 16 + y * 4 + x] + start_block_x;
				pos_sub_l[1] = pos_warp[1] + D_pppDFBlock_y1[z * 16 + y * 4 + x] + start_block_y;
				pos_sub_l[2] = pos_warp[2] + D_pppDFBlock_z1[z * 16 + y * 4 + x] + start_block_z;

				if (pos_sub_l[0]<0 || pos_sub_l[0]>gs_ori0 - 1 ||
					pos_sub_l[1]<0 || pos_sub_l[1]>gs_ori1 - 1 ||
					pos_sub_l[2]<0 || pos_sub_l[2]>gs_ori2 - 1)
				{
					for (V3DLONG c = 0; c < D_sz_img_sub3; c++)
						D_p_img_warp_4d[pos_warp[2] * D_sz_img_sub1 * D_sz_img_sub0 + pos_warp[1] * D_sz_img_sub0 + pos_warp[0]] = 0;
					continue;
				}
				if (pos_sub[0]<0 || pos_sub[0]>D_sz_img_read_sub0 - 1 ||
					pos_sub[1]<0 || pos_sub[1]>D_sz_img_read_sub1 - 1 ||
					pos_sub[2]<0 || pos_sub[2]>D_sz_img_read_sub2 - 1)
				{
					for (V3DLONG c = 0; c < D_sz_img_sub3; c++)
						D_p_img_warp_4d[pos_warp[2] * D_sz_img_sub1 * D_sz_img_sub0 + pos_warp[1] * D_sz_img_sub0 + pos_warp[0]] = 0;
					continue;
				}


				long long x_s, x_b, y_s, y_b, z_s, z_b;
				x_s = floor(pos_sub[0]);            x_b = ceil(pos_sub[0]);
				y_s = floor(pos_sub[1]);            y_b = ceil(pos_sub[1]);
				z_s = floor(pos_sub[2]);            z_b = ceil(pos_sub[2]);
				double l_w, r_w, t_w, b_w;
				l_w = 1.0 - (pos_sub[0] - x_s);  r_w = 1.0 - l_w;
				t_w = 1.0 - (pos_sub[1] - y_s);  b_w = 1.0 - t_w;
				double u_w, d_w;
				u_w = 1.0 - (pos_sub[2] - z_s);  d_w = 1.0 - u_w;


				double higher_slice;
				higher_slice = t_w*(l_w*D_p_img_sub_4d[z_s * D_sz_img_read_sub1 * D_sz_img_read_sub0 + y_s * D_sz_img_read_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s * D_sz_img_read_sub1 * D_sz_img_read_sub0 + y_s * D_sz_img_read_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_s * D_sz_img_read_sub1 * D_sz_img_read_sub0 + y_b * D_sz_img_read_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s * D_sz_img_read_sub1 * D_sz_img_read_sub0 + y_b*D_sz_img_read_sub0 + x_b]);
				double lower_slice;
				lower_slice = t_w*(l_w*D_p_img_sub_4d[z_b * D_sz_img_read_sub1 * D_sz_img_read_sub0 + y_s * D_sz_img_read_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b * D_sz_img_read_sub1 * D_sz_img_read_sub0 + y_s * D_sz_img_read_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_b * D_sz_img_read_sub1 * D_sz_img_read_sub0 + y_b * D_sz_img_read_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b * D_sz_img_read_sub1 * D_sz_img_read_sub0 + y_b*D_sz_img_read_sub0 + x_b]);
				double intval = (u_w*higher_slice + d_w*lower_slice + 0.5);

				D_p_img_warp_4d[pos_warp[2] * D_sz_img_sub1 * D_sz_img_sub0 + pos_warp[1] * D_sz_img_sub0 + pos_warp[0]] = intval;


			}
}


__global__ void get_Displacement_brightness_stps(const V3DLONG sz_gridwnd, const int k, const V3DLONG gsz1, const V3DLONG gsz0, float * D_pppSubDF_x, float * D_pppSubDF_y, float * D_pppSubDF_z,
	float *D_x_bsplinebasis, float *D_pppDFBlock_x, float *D_pppDFBlock_y, float *D_pppDFBlock_z, const int szBlock_x, const int szBlock_y, const int szBlock_z, const int D_sz_img_sub0,
	const int D_sz_img_sub1, const int D_sz_img_sub2, const int D_sz_img_sub3, unsigned char *D_p_img_sub_4d, unsigned char *D_p_img_warp_4d, const int i_interpmethod_img, const int D_sz_img_ori_sub0,
	const int D_sz_img_ori_sub1, const int D_sz_img_ori_sub2, const int D_sz_img_ori_sub3)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;
//	printf("grid[%d %d]\n", row, col);
	if (row >= gsz1 - 1 - 2 || col >= gsz0 - 1 - 2)return;
	//printf("mark221");
	float x1D_gridblock[192];
	int ind = 1;
	for (int a = k; a<k + 4; a++)
		for (int c = col; c<col + 4; c++)
			for (int b = row; b<row + 4; b++)
			{
				x1D_gridblock[(ind - 1) * 3] = D_pppSubDF_x[a*gsz1*gsz0 + b*gsz0 + c];
				x1D_gridblock[(ind - 1) * 3 + 1] = D_pppSubDF_y[a*gsz1*gsz0 + b*gsz0 + c];
				x1D_gridblock[(ind - 1) * 3 + 2] = D_pppSubDF_z[a*gsz1*gsz0 + b*gsz0 + c];
				ind++;
			}
	//	printf("grid[%d %d],x:%f y:%f z:%f\n", row, col,x1D_gridblock[0], x1D_gridblock[1], x1D_gridblock[2]);

	float  x1D_gridblock_int[4 * 4 * 4 * 3];


	MUL(D_x_bsplinebasis, x1D_gridblock, x1D_gridblock_int);
	//printf("grid[%d %d],%f %f %f\n", row, col, x1D_gridblock_int[2], x1D_gridblock_int[1], x1D_gridblock_int[0]);
	//printf("ind:%d\n", ind);
	//printf("grid[%d %d %d],%f %f %f\n", k,row, col, x1D_gridblock_int[2], x1D_gridblock_int[1], x1D_gridblock_int[0]);
	int idx = 1;
	float D_pppDFBlock_x1[4 * 4 * 4]; float D_pppDFBlock_y1[4 * 4 * 4]; float D_pppDFBlock_z1[4 * 4 * 4];
	//printf("mark22");
	for (long zz = 0; zz<sz_gridwnd; zz++)
		for (long xx = 0; xx<sz_gridwnd; xx++)
			for (long yy = 0; yy<sz_gridwnd; yy++)
			{
				D_pppDFBlock_x1[zz * 16 + yy * 4 + xx] = x1D_gridblock_int[(idx - 1) * 3];
				D_pppDFBlock_y1[zz * 16 + yy * 4 + xx] = x1D_gridblock_int[(idx - 1) * 3 + 1];
				D_pppDFBlock_z1[zz * 16 + yy * 4 + xx] = x1D_gridblock_int[(idx - 1) * 3 + 2];
				idx++;
			}
	//printf("grid[%d %d %d],%f %f %f\n", k, row, col, D_pppDFBlock_x1[0], D_pppDFBlock_y1[0], D_pppDFBlock_z1[0]);
	//printf("mar222");
	int start_x, start_y, start_z;
	start_x = col*szBlock_x;
	start_y = row*szBlock_y;
	start_z = k*szBlock_z;
	//printf("mar333");
	for (int z = 0; z < szBlock_z; z++)
		for (int y = 0; y < szBlock_y; y++)
			for (int x = 0; x < szBlock_x; x++)
			{
				//printf("mar222");
				int pos_warp[3];
				pos_warp[0] = start_x + x;
				pos_warp[1] = start_y + y;
				pos_warp[2] = start_z + z;
				//printf("x:%d  y:%d  z:%d idx:%d  grid[%d %d %d],%d %d %d\n",x,y,z,x+y*4+z*16, k, row, col, pos_warp[0], pos_warp[1], pos_warp[2]);


				if (pos_warp[0] >= D_sz_img_sub0 || pos_warp[1] >= D_sz_img_sub1 || pos_warp[2] >= D_sz_img_sub2)
					continue;

				double pos_sub[3];
				pos_sub[0] = pos_warp[0] + D_pppDFBlock_x1[z * 16 + y * 4 + x];
				pos_sub[1] = pos_warp[1] + D_pppDFBlock_y1[z * 16 + y * 4 + x];
				pos_sub[2] = pos_warp[2] + D_pppDFBlock_z1[z * 16 + y * 4 + x];
				//printf("x:%d  y:%d  z:%d idx:%d  grid[%d %d %d],%f %f %f\n", x, y, z, x + y * 4 + z * 16, k, row, col, pos_sub[0], pos_sub[1], pos_sub[2]);

				//if (pos_sub[0]<0 || pos_sub[0]>D_sz_img_sub0 - 1 || pos_sub[1]<0 || pos_sub[1]>D_sz_img_sub1 - 1 || pos_sub[2]<0 || pos_sub[2]>D_sz_img_sub2 - 1)
				if (pos_sub[0]<0 || pos_sub[0]>D_sz_img_ori_sub0 - 1 || pos_sub[1]<0 || pos_sub[1]>D_sz_img_ori_sub1 - 1 || pos_sub[2]<0 || pos_sub[2]>D_sz_img_ori_sub2 - 1)
				{
					for (V3DLONG c = 0; c < D_sz_img_sub3; c++)
						D_p_img_warp_4d[pos_warp[2] * D_sz_img_sub1 * D_sz_img_sub0 + pos_warp[1] * D_sz_img_sub0 + pos_warp[0]] = 0;
					continue;
				}


				//find 8 neighor pixels boundary

				int x_s, x_b, y_s, y_b, z_s, z_b;
				x_s = myfloor(pos_sub[0]);            x_b = myceil(pos_sub[0]);
				y_s = myfloor(pos_sub[1]);            y_b = myceil(pos_sub[1]);
				z_s = myfloor(pos_sub[2]);            z_b = myceil(pos_sub[2]);
				//printf("x:%d y:%d z:%d idx:%d grid[%d %d %d],%f %f %f,down:%d %d %d up:%d %d %d\n", x, y, z, x + y * 4 + z * 16, k, row, col, pos_sub[0], pos_sub[1], pos_sub[2], x_s, y_s, z_s, x_b, y_b, z_b);

				//compute weight for left and right, top and  bottom -- 4 neighbor pixel's weight in a slice
				double l_w, r_w, t_w, b_w;
				l_w = 1.0 - (pos_sub[0] - x_s);  r_w = 1.0 - l_w;
				t_w = 1.0 - (pos_sub[1] - y_s);  b_w = 1.0 - t_w;
				double u_w, d_w;
				u_w = 1.0 - (pos_sub[2] - z_s);  d_w = 1.0 - u_w;
				//printf("x:%d y:%d z:%d idx:%d grid[%d %d %d],%f %f %f,l_w:%f r_w:%f t_w:%f b_w:%f u_w:%f d_w:%f\n", x, y, z, x + y * 4 + z * 16, k, row, col, pos_sub[0], pos_sub[1], pos_sub[2], l_w, r_w, t_w, b_w, u_w, d_w);
				//printf("x:%d y:%d z:%d idx:%d grid[%d %d %d],%f %f %f,down:%d %d %d up:%d %d %d\n", x, y, z, x + y * 4 + z * 16, k, row, col, pos_sub[0], pos_sub[1], pos_sub[2], x_s, y_s, z_s, x_b, y_b, z_b);
				//printf("l_w:%f r_w:%f t_w:%f b_w:%f u_w:%f d_w:%f %f %f %f\n", l_w, r_w, t_w, b_w, u_w, d_w, pos_sub[0], pos_sub[1], pos_sub[2]);

				double higher_slice;
				//higher_slice = t_w*(l_w*D_p_img_sub_4d[z_s*D_sz_img_sub1 * D_sz_img_sub0 + y_s*D_sz_img_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s*D_sz_img_sub1 * D_sz_img_sub0 + y_s*D_sz_img_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_s*D_sz_img_sub1 * D_sz_img_sub0 + y_b*D_sz_img_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s*D_sz_img_sub1 * D_sz_img_sub0 + y_b*D_sz_img_sub0 + x_b]);
				higher_slice = t_w*(l_w*D_p_img_sub_4d[z_s * D_sz_img_ori_sub1 * D_sz_img_ori_sub0 + y_s * D_sz_img_ori_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s * D_sz_img_ori_sub1 * D_sz_img_ori_sub0 + y_s * D_sz_img_ori_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_s * D_sz_img_ori_sub1 * D_sz_img_ori_sub0 + y_b * D_sz_img_ori_sub0 + x_s] + r_w*D_p_img_sub_4d[z_s * D_sz_img_ori_sub1 * D_sz_img_ori_sub0 + y_b*D_sz_img_ori_sub0 + x_b]);
				double lower_slice;
				//lower_slice = t_w*(l_w*D_p_img_sub_4d[z_b*D_sz_img_sub1 * D_sz_img_sub0 + y_s*D_sz_img_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b*D_sz_img_sub1 * D_sz_img_sub0 + y_s*D_sz_img_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_b*D_sz_img_sub1 * D_sz_img_sub0 + y_b*D_sz_img_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b*D_sz_img_sub1 * D_sz_img_sub0 + y_b*D_sz_img_sub0 + x_b]);
				lower_slice = t_w*(l_w*D_p_img_sub_4d[z_b * D_sz_img_ori_sub1 * D_sz_img_ori_sub0 + y_s * D_sz_img_ori_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b * D_sz_img_ori_sub1 * D_sz_img_ori_sub0 + y_s * D_sz_img_ori_sub0 + x_b]) + b_w*(l_w*D_p_img_sub_4d[z_b * D_sz_img_ori_sub1 * D_sz_img_ori_sub0 + y_b * D_sz_img_ori_sub0 + x_s] + r_w*D_p_img_sub_4d[z_b * D_sz_img_ori_sub1 * D_sz_img_ori_sub0 + y_b*D_sz_img_ori_sub0 + x_b]);
				double intval = (u_w*higher_slice + d_w*lower_slice + 0.5);

				D_p_img_warp_4d[pos_warp[2] * D_sz_img_sub1 * D_sz_img_sub0 + pos_warp[1] * D_sz_img_sub0 + pos_warp[0]] = intval;





			}
}


__global__ void get_Displacement_brightness_nn(const V3DLONG sz_gridwnd, const int k, const V3DLONG gsz1, const V3DLONG gsz0, float * D_pppSubDF_x, float * D_pppSubDF_y, float * D_pppSubDF_z,
	float *D_x_bsplinebasis, float *D_pppDFBlock_x, float *D_pppDFBlock_y, float *D_pppDFBlock_z, const int szBlock_x, const int szBlock_y, const int szBlock_z, const int D_sz_img_sub0,
	const int D_sz_img_sub1, const int D_sz_img_sub2, const int D_sz_img_sub3, unsigned char *D_p_img_sub_4d, unsigned char *D_p_img_warp_4d, const int i_interpmethod_img, const int D_sz_img_ori_sub0,
	const int D_sz_img_ori_sub1, const int D_sz_img_ori_sub2, const int D_sz_img_ori_sub3)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("grid[%d %d]\n", row, col);
	if (row >= gsz1 - 1 - 2 || col >= gsz0 - 1 - 2)return;
	//printf("mark221");
	float x1D_gridblock[192];
	int ind = 1;
	for (int a = k; a<k + 4; a++)
		for (int c = col; c<col + 4; c++)
			for (int b = row; b<row + 4; b++)
			{
				x1D_gridblock[(ind - 1) * 3] = D_pppSubDF_x[a*gsz1*gsz0 + b*gsz0 + c];
				x1D_gridblock[(ind - 1) * 3 + 1] = D_pppSubDF_y[a*gsz1*gsz0 + b*gsz0 + c];
				x1D_gridblock[(ind - 1) * 3 + 2] = D_pppSubDF_z[a*gsz1*gsz0 + b*gsz0 + c];
				ind++;
			}
	//	printf("grid[%d %d],x:%f y:%f z:%f\n", row, col,x1D_gridblock[0], x1D_gridblock[1], x1D_gridblock[2]);

	float  x1D_gridblock_int[4 * 4 * 4 * 3];


	MUL(D_x_bsplinebasis, x1D_gridblock, x1D_gridblock_int);
	//printf("grid[%d %d],%f %f %f\n", row, col, x1D_gridblock_int[2], x1D_gridblock_int[1], x1D_gridblock_int[0]);
	//printf("ind:%d\n", ind);
	//printf("grid[%d %d %d],%f %f %f\n", k,row, col, x1D_gridblock_int[2], x1D_gridblock_int[1], x1D_gridblock_int[0]);
	int idx = 1;
	float D_pppDFBlock_x1[4 * 4 * 4]; float D_pppDFBlock_y1[4 * 4 * 4]; float D_pppDFBlock_z1[4 * 4 * 4];
	//printf("mark22");
	for (long zz = 0; zz<sz_gridwnd; zz++)
		for (long xx = 0; xx<sz_gridwnd; xx++)
			for (long yy = 0; yy<sz_gridwnd; yy++)
			{
				D_pppDFBlock_x1[zz * 16 + yy * 4 + xx] = x1D_gridblock_int[(idx - 1) * 3];
				D_pppDFBlock_y1[zz * 16 + yy * 4 + xx] = x1D_gridblock_int[(idx - 1) * 3 + 1];
				D_pppDFBlock_z1[zz * 16 + yy * 4 + xx] = x1D_gridblock_int[(idx - 1) * 3 + 2];
				idx++;
			}
	//printf("grid[%d %d %d],%f %f %f\n", k, row, col, D_pppDFBlock_x1[0], D_pppDFBlock_y1[0], D_pppDFBlock_z1[0]);
	//printf("mar222");
	int start_x, start_y, start_z;
	start_x = col*szBlock_x;
	start_y = row*szBlock_y;
	start_z = k*szBlock_z;
	//printf("mar333");
	for (int z = 0; z < szBlock_z; z++)
		for (int y = 0; y < szBlock_y; y++)
			for (int x = 0; x < szBlock_x; x++)
			{
				//printf("mar222");
				int pos_warp[3];
				pos_warp[0] = start_x + x;
				pos_warp[1] = start_y + y;
				pos_warp[2] = start_z + z;
				//printf("x:%d  y:%d  z:%d idx:%d  grid[%d %d %d],%d %d %d\n",x,y,z,x+y*4+z*16, k, row, col, pos_warp[0], pos_warp[1], pos_warp[2]);


				if (pos_warp[0] >= D_sz_img_sub0 || pos_warp[1] >= D_sz_img_sub1 || pos_warp[2] >= D_sz_img_sub2)
					continue;

				double pos_sub[3];
				pos_sub[0] = pos_warp[0] + D_pppDFBlock_x1[z * 16 + y * 4 + x];
				pos_sub[1] = pos_warp[1] + D_pppDFBlock_y1[z * 16 + y * 4 + x];
				pos_sub[2] = pos_warp[2] + D_pppDFBlock_z1[z * 16 + y * 4 + x];
				//printf("x:%d  y:%d  z:%d idx:%d  grid[%d %d %d],%f %f %f\n", x, y, z, x + y * 4 + z * 16, k, row, col, pos_sub[0], pos_sub[1], pos_sub[2]);

				//if (pos_sub[0]<0 || pos_sub[0]>D_sz_img_sub0 - 1 || pos_sub[1]<0 || pos_sub[1]>D_sz_img_sub1 - 1 || pos_sub[2]<0 || pos_sub[2]>D_sz_img_sub2 - 1)
				if (pos_sub[0]<0 || pos_sub[0]>D_sz_img_ori_sub0 - 1 || pos_sub[1]<0 || pos_sub[1]>D_sz_img_ori_sub1 - 1 || pos_sub[2]<0 || pos_sub[2]>D_sz_img_ori_sub2 - 1)
				{
					for (V3DLONG c = 0; c < D_sz_img_sub3; c++)
						D_p_img_warp_4d[pos_warp[2] * D_sz_img_sub1 * D_sz_img_sub0 + pos_warp[1] * D_sz_img_sub0 + pos_warp[0]] = 0;
					continue;
				}

				///nearest neighbor interpolate

				long long pos_sub_nn[3];

				pos_sub_nn[0] = pos_sub[0] + 0.5;
				pos_sub_nn[0] = pos_sub_nn[0]<D_sz_img_ori_sub0 ? pos_sub_nn[0] : D_sz_img_ori_sub0 - 1;

				pos_sub_nn[1] = pos_sub[1] + 0.5;
				pos_sub_nn[1] = pos_sub_nn[1]<D_sz_img_ori_sub1 ? pos_sub_nn[1] : D_sz_img_ori_sub1 - 1;

				pos_sub_nn[2] = pos_sub[2] + 0.5;
				pos_sub_nn[2] = pos_sub_nn[2]<D_sz_img_ori_sub2 ? pos_sub_nn[2] : D_sz_img_ori_sub2 - 1;

				D_p_img_warp_4d[pos_warp[2] * D_sz_img_sub1 * D_sz_img_sub0 + pos_warp[1] * D_sz_img_sub0 + pos_warp[0]] = D_p_img_sub_4d[pos_sub_nn[2] * D_sz_img_ori_sub1 * D_sz_img_ori_sub0 + pos_sub_nn[1] * D_sz_img_ori_sub0 + pos_sub_nn[0]];




			}
}

__global__ void get_Displacement_brightness_sort(const V3DLONG sz_gridwnd, const int k, const V3DLONG gsz1, const V3DLONG gsz0, float * D_pppSubDF_x, float * D_pppSubDF_y, float * D_pppSubDF_z,
	float *D_x_bsplinebasis, float *D_pppDFBlock_x, float *D_pppDFBlock_y, float *D_pppDFBlock_z, const int szBlock_x, const int szBlock_y, const int szBlock_z, const int D_sz_img_sub0,
	const int D_sz_img_sub1, const int D_sz_img_sub2, const int D_sz_img_sub3, const int i_interpmethod_img, const long long start_block_x, const long long start_block_y, const long long start_block_z,
	const long long gs_ori2, const long long gs_ori1, const long long gs_ori0, float *D_sort_x, float *D_sort_y, float *D_sort_z)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("grid[%d %d]\n", row, col);
	if (row >= gsz1 - 1 - 2 || col >= gsz0 - 1 - 2)return;
	//printf("mark221");
	float x1D_gridblock[192];
	int ind = 1;
	for (int a = k; a<k + 4; a++)
		for (int c = col; c<col + 4; c++)
			for (int b = row; b<row + 4; b++)
			{
				x1D_gridblock[(ind - 1) * 3] = D_pppSubDF_x[a*gsz1*gsz0 + b*gsz0 + c];
				x1D_gridblock[(ind - 1) * 3 + 1] = D_pppSubDF_y[a*gsz1*gsz0 + b*gsz0 + c];
				x1D_gridblock[(ind - 1) * 3 + 2] = D_pppSubDF_z[a*gsz1*gsz0 + b*gsz0 + c];
				ind++;
			}
	//	printf("grid[%d %d],x:%f y:%f z:%f\n", row, col,x1D_gridblock[0], x1D_gridblock[1], x1D_gridblock[2]);

	float  x1D_gridblock_int[4 * 4 * 4 * 3];


	MUL(D_x_bsplinebasis, x1D_gridblock, x1D_gridblock_int);
	//printf("grid[%d %d],%f %f %f\n", row, col, x1D_gridblock_int[2], x1D_gridblock_int[1], x1D_gridblock_int[0]);
	//printf("ind:%d\n", ind);
	//printf("grid[%d %d %d],%f %f %f\n", k,row, col, x1D_gridblock_int[2], x1D_gridblock_int[1], x1D_gridblock_int[0]);
	int idx = 1;
	float D_pppDFBlock_x1[4 * 4 * 4]; float D_pppDFBlock_y1[4 * 4 * 4]; float D_pppDFBlock_z1[4 * 4 * 4];
	//printf("mark22");
	for (long zz = 0; zz<sz_gridwnd; zz++)
		for (long xx = 0; xx<sz_gridwnd; xx++)
			for (long yy = 0; yy<sz_gridwnd; yy++)
			{
				D_pppDFBlock_x1[zz * 16 + yy * 4 + xx] = x1D_gridblock_int[(idx - 1) * 3];
				D_pppDFBlock_y1[zz * 16 + yy * 4 + xx] = x1D_gridblock_int[(idx - 1) * 3 + 1];
				D_pppDFBlock_z1[zz * 16 + yy * 4 + xx] = x1D_gridblock_int[(idx - 1) * 3 + 2];
				idx++;
			}
	//printf("grid[%d %d %d],%f %f %f\n", k, row, col, D_pppDFBlock_x1[0], D_pppDFBlock_y1[0], D_pppDFBlock_z1[0]);
	//printf("mar222");
	long long start_x, start_y, start_z;
	start_x = col*szBlock_x;
	start_y = row*szBlock_y;
	start_z = k*szBlock_z;
	//printf("mar333");
	for (int z = 0; z < szBlock_z; z++)
		for (int y = 0; y < szBlock_y; y++)
			for (int x = 0; x < szBlock_x; x++)
			{
				//printf("mar222");
				long long pos_warp[3];
				pos_warp[0] = start_x + x;
				pos_warp[1] = start_y + y;
				pos_warp[2] = start_z + z;
				//printf("x:%d  y:%d  z:%d idx:%d  grid[%d %d %d],%d %d %d\n",x,y,z,x+y*4+z*16, k, row, col, pos_warp[0], pos_warp[1], pos_warp[2]);


				if (pos_warp[0] >= D_sz_img_sub0 || pos_warp[1] >= D_sz_img_sub1 || pos_warp[2] >= D_sz_img_sub2)
					continue;

				double pos_sub[3], pos_sub_l[3];
				//pos_sub[0] = pos_warp[0] + D_pppDFBlock_x1[z * 16 + y * 4 + x] + start_block_x - x_read_offset;
				//pos_sub[1] = pos_warp[1] + D_pppDFBlock_y1[z * 16 + y * 4 + x] + start_block_y - y_read_offset;
				//pos_sub[2] = pos_warp[2] + D_pppDFBlock_z1[z * 16 + y * 4 + x] + start_block_z - z_read_offset;

				pos_sub_l[0] = pos_warp[0] + D_pppDFBlock_x1[z * 16 + y * 4 + x] + start_block_x;
				pos_sub_l[1] = pos_warp[1] + D_pppDFBlock_y1[z * 16 + y * 4 + x] + start_block_y;
				pos_sub_l[2] = pos_warp[2] + D_pppDFBlock_z1[z * 16 + y * 4 + x] + start_block_z;
				//printf("x:%d  y:%d  z:%d idx:%d  grid[%d %d %d],%f %f %f\n", x, y, z, x + y * 4 + z * 16, k, row, col, pos_sub[0], pos_sub[1], pos_sub[2]);

				//if (pos_sub[0]<0 || pos_sub[0]>D_sz_img_sub0 - 1 || pos_sub[1]<0 || pos_sub[1]>D_sz_img_sub1 - 1 || pos_sub[2]<0 || pos_sub[2]>D_sz_img_sub2 - 1)
				/*if (pos_sub_l[0]<0 || pos_sub_l[0]>gs_ori0 - 1 ||
				pos_sub_l[1]<0 || pos_sub_l[1]>gs_ori1 - 1 ||
				pos_sub_l[2]<0 || pos_sub_l[2]>gs_ori2 - 1)
				{
				for (V3DLONG c = 0; c < D_sz_img_sub3; c++)
				D_p_img_warp_4d[pos_warp[2] * D_sz_img_sub1 * D_sz_img_sub0 + pos_warp[1] * D_sz_img_sub0 + pos_warp[0]] = 0;
				continue;
				}*/
				if (pos_sub_l[0] >= 0 && pos_sub_l[0] <= gs_ori0 - 1)
				{
					D_sort_x[pos_warp[0]] = pos_sub_l[0];
				}
				if (pos_sub_l[1] >= 0 && pos_sub_l[1] <= gs_ori1 - 1)
				{
					D_sort_y[pos_warp[1]] = pos_sub_l[1];
				}
				if (pos_sub_l[2] >= 0 && pos_sub_l[2] <= gs_ori2 - 1)
				{
					D_sort_z[pos_warp[2]] = pos_sub_l[2];
				}
				//D_sort_x[pos_warp[0]] = pos_sub_l[0];
				//D_sort_y[pos_warp[1]] = pos_sub_l[1];
				//D_sort_z[pos_warp[2]] = pos_sub_l[2];



				/*if (pos_sub[0]<0 || pos_sub[0]>D_sz_img_read_sub0 - 1 ||
				pos_sub[1]<0 || pos_sub[1]>D_sz_img_read_sub1 - 1 ||
				pos_sub[2]<0 || pos_sub[2]>D_sz_img_read_sub2 - 1)
				{
				for (V3DLONG c = 0; c < D_sz_img_sub3; c++)
				D_p_img_warp_4d[pos_warp[2] * D_sz_img_sub1 * D_sz_img_sub0 + pos_warp[1] * D_sz_img_sub0 + pos_warp[0]] = 0;
				continue;
				}*/
			}
}






extern "C" bool gpu_interpolation(const int gsz2, const int gsz1, const int gsz0, DisplaceFieldF3D ***&pppSubDF, const Matrix &x_bsplinebasis, const V3DLONG sz_gridwnd, DisplaceFieldF3D ***&pppDFBlock,
	unsigned char ****&p_img_sub_4d, const V3DLONG *sz_img_sub, const V3DLONG szBlock_x, const V3DLONG szBlock_y, const V3DLONG szBlock_z, const int i_interpmethod_img, unsigned char ****&p_img_warp_4d,
	const V3DLONG *sz_img_sub_read, const unsigned char *p_img_sub, unsigned char *p_img_warp, const long long start_block_x, const long long start_block_y, const long long start_block_z,
	const long long x_read_offset, const long long y_read_offset, const long long z_read_offset, const long long gs_ori2, const long long gs_ori1, const long long gs_ori0)
{
	float *H_pppSubDF_x, *H_pppSubDF_y, *H_pppSubDF_z, *H_pppDFBlock_x, *H_pppDFBlock_y, *H_pppDFBlock_z;
	float *H_x_bsplinebasis, *D_x_bsplinebasis;
	float *D_pppSubDF_x, *D_pppSubDF_y, *D_pppSubDF_z, *D_pppDFBlock_x, *D_pppDFBlock_y, *D_pppDFBlock_z;
	H_pppSubDF_x = (float*)malloc(gsz2 * gsz1 *gsz0 * sizeof(float));
	H_pppSubDF_y = (float*)malloc(gsz2 * gsz1 *gsz0 * sizeof(float));
	H_pppSubDF_z = (float*)malloc(gsz2 * gsz1 *gsz0 * sizeof(float));

	H_x_bsplinebasis = (float*)malloc(x_bsplinebasis.nrows() * x_bsplinebasis.ncols() * sizeof(float));
	H_pppDFBlock_x = (float*)malloc(sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));
	H_pppDFBlock_y = (float*)malloc(sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));
	H_pppDFBlock_z = (float*)malloc(sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));

	hipMalloc((void**)&D_pppSubDF_x, gsz2 * gsz1 *gsz0 * sizeof(float));
	hipMalloc((void**)&D_pppSubDF_y, gsz2 * gsz1 *gsz0 * sizeof(float));
	hipMalloc((void**)&D_pppSubDF_z, gsz2 * gsz1 *gsz0 * sizeof(float));
	hipMalloc((void**)&D_x_bsplinebasis, x_bsplinebasis.nrows() * x_bsplinebasis.ncols() * sizeof(float));
	hipMalloc((void**)&D_pppDFBlock_x, sz_gridwnd * sz_gridwnd *sz_gridwnd* sizeof(float));
	hipMalloc((void**)&D_pppDFBlock_y, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));
	hipMalloc((void**)&D_pppDFBlock_z, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));


	unsigned char *D_p_img_sub_4d, *D_p_img_warp_4d;

	hipMalloc((void**)&D_p_img_sub_4d, sz_img_sub_read[0] * sz_img_sub_read[1] * sz_img_sub_read[2] * sz_img_sub_read[3] * sizeof(unsigned char));
	hipMalloc((void**)&D_p_img_warp_4d, sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(unsigned char));


	int aa = sz_img_sub[0]; int bb = sz_img_sub[1]; int cc = sz_img_sub[2]; int dd = sz_img_sub[3];
	int aa_read = sz_img_sub_read[0]; int bb_read = sz_img_sub_read[1]; int cc_read = sz_img_sub_read[2]; int dd_read = sz_img_sub_read[3];
	for (V3DLONG a = 0; a < gsz2; a++)
	{
		for (V3DLONG b = 0; b < gsz1; b++)
		{
			for (V3DLONG c = 0; c < gsz0; c++)
			{

				H_pppSubDF_x[a*gsz1*gsz0 + b*gsz0 + c] = pppSubDF[a][b][c].sx;
				H_pppSubDF_y[a*gsz1*gsz0 + b*gsz0 + c] = pppSubDF[a][b][c].sy;
				H_pppSubDF_z[a*gsz1*gsz0 + b*gsz0 + c] = pppSubDF[a][b][c].sz;

			}
		}
	}

	for (int i = 0; i < x_bsplinebasis.nrows(); i++)
	{
		for (int j = 0; j < x_bsplinebasis.ncols(); j++)
		{

			H_x_bsplinebasis[i * x_bsplinebasis.ncols() + j] = x_bsplinebasis(i + 1, j + 1);

		}
	}


	HANDLE_ERROR(hipMemcpy(D_pppSubDF_x, H_pppSubDF_x, gsz2 * gsz1 *gsz0 * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppSubDF_y, H_pppSubDF_y, gsz2 * gsz1 *gsz0 * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppSubDF_z, H_pppSubDF_z, gsz2 * gsz1 *gsz0 * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_x_bsplinebasis, H_x_bsplinebasis, x_bsplinebasis.nrows() * x_bsplinebasis.ncols() * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppDFBlock_x, H_pppDFBlock_x, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppDFBlock_y, H_pppDFBlock_y, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppDFBlock_z, H_pppDFBlock_z, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float), hipMemcpyHostToDevice));


	HANDLE_ERROR(hipMemcpy(D_p_img_sub_4d, p_img_sub, sz_img_sub_read[0] * sz_img_sub_read[1] * sz_img_sub_read[2] * sz_img_sub_read[3] * sizeof(unsigned char), hipMemcpyHostToDevice));





	for (int k = 0; k < gsz2 - 1 - 2; k++)
	{
		dim3 grid((gsz0 - 1 - 2 + 16 - 1) / 16, (gsz1 - 1 - 2 + 16 - 1) / 16);
		dim3 block(16, 16);
		get_Displacement_brightness << <grid, block >> >(sz_gridwnd, k, gsz1, gsz0, D_pppSubDF_x, D_pppSubDF_y, D_pppSubDF_z, D_x_bsplinebasis, szBlock_x, szBlock_y, szBlock_z, aa, bb, cc, dd,
                                                         D_p_img_sub_4d, D_p_img_warp_4d, aa_read, bb_read, cc_read, start_block_x,start_block_y, start_block_z,
                                                         x_read_offset, y_read_offset, z_read_offset, gs_ori2, gs_ori1, gs_ori0);
//        HANDLE_ERROR(hipGetLastError());
		hipDeviceSynchronize();
	}


	HANDLE_ERROR(hipMemcpy(p_img_warp, D_p_img_warp_4d, sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(unsigned char), hipMemcpyDeviceToHost));



	free(H_pppSubDF_x);
	free(H_pppSubDF_y);
	free(H_pppSubDF_z);

	free(H_x_bsplinebasis);

	free(H_pppDFBlock_x);
	free(H_pppDFBlock_y);
	free(H_pppDFBlock_z);

	hipFree(D_pppSubDF_x); hipFree(D_pppSubDF_y); hipFree(D_pppSubDF_z);
	hipFree(D_pppDFBlock_x); hipFree(D_pppDFBlock_y); hipFree(D_pppDFBlock_z);
	hipFree(D_x_bsplinebasis); hipFree(D_p_img_sub_4d); hipFree(D_p_img_warp_4d);
	HANDLE_ERROR(hipDeviceReset());
	return true;
}


extern "C" bool gpu_interpolation_sort(const int gsz2, const int gsz1, const int gsz0, DisplaceFieldF3D ***&pppSubDF, const Matrix &x_bsplinebasis, const V3DLONG sz_gridwnd, DisplaceFieldF3D ***&pppDFBlock,
	const V3DLONG *sz_img_sub, const V3DLONG szBlock_x, const V3DLONG szBlock_y, const V3DLONG szBlock_z, const int i_interpmethod_img, const long long start_block_x, const long long start_block_y,
	const long long start_block_z, const long long gs_ori2, const long long gs_ori1, const long long gs_ori0, float * &sort_x, float * &sort_y, float * &sort_z)
{
	//hipEvent_t   start, stop;
	//HANDLE_ERROR(hipEventCreate(&start));
	//HANDLE_ERROR(hipEventCreate(&stop));
	float *H_pppSubDF_x, *H_pppSubDF_y, *H_pppSubDF_z, *H_pppDFBlock_x, *H_pppDFBlock_y, *H_pppDFBlock_z;
	float *H_x_bsplinebasis, *D_x_bsplinebasis;
	float *D_pppSubDF_x, *D_pppSubDF_y, *D_pppSubDF_z, *D_pppDFBlock_x, *D_pppDFBlock_y, *D_pppDFBlock_z;
	float *D_sort_x, *D_sort_y, *D_sort_z;
	H_pppSubDF_x = (float*)malloc(gsz2 * gsz1 *gsz0 * sizeof(float));
	H_pppSubDF_y = (float*)malloc(gsz2 * gsz1 *gsz0 * sizeof(float));
	H_pppSubDF_z = (float*)malloc(gsz2 * gsz1 *gsz0 * sizeof(float));

	H_x_bsplinebasis = (float*)malloc(x_bsplinebasis.nrows() * x_bsplinebasis.ncols() * sizeof(float));
	H_pppDFBlock_x = (float*)malloc(sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));
	H_pppDFBlock_y = (float*)malloc(sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));
	H_pppDFBlock_z = (float*)malloc(sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));

	hipMalloc((void**)&D_pppSubDF_x, gsz2 * gsz1 *gsz0 * sizeof(float));
	hipMalloc((void**)&D_pppSubDF_y, gsz2 * gsz1 *gsz0 * sizeof(float));
	hipMalloc((void**)&D_pppSubDF_z, gsz2 * gsz1 *gsz0 * sizeof(float));
	hipMalloc((void**)&D_x_bsplinebasis, x_bsplinebasis.nrows() * x_bsplinebasis.ncols() * sizeof(float));
	hipMalloc((void**)&D_pppDFBlock_x, sz_gridwnd * sz_gridwnd *sz_gridwnd* sizeof(float));
	hipMalloc((void**)&D_pppDFBlock_y, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));
	hipMalloc((void**)&D_pppDFBlock_z, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));

	hipMalloc((void**)&D_sort_x, sz_img_sub[0] * sizeof(float));
	hipMalloc((void**)&D_sort_y, sz_img_sub[1] * sizeof(float));
	hipMalloc((void**)&D_sort_z, sz_img_sub[2] * sizeof(float));


	unsigned char *D_p_img_sub_4d, *D_p_img_warp_4d;
	//H_p_img_sub_4d = (float*)malloc(2 * sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float));
	//H_p_img_sub_4d = (float*)malloc(sz_img_sub_ori[0] * sz_img_sub_ori[1] * sz_img_sub_ori[2] * sz_img_sub[3] * sizeof(float));
	//H_p_img_warp_4d = (float*)malloc(sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float));
	//H_sz_img_sub = (float*)malloc(4 * sizeof(float));
	//hipMalloc((void**)&D_p_img_sub_4d, 2 * sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float));
	//hipMalloc((void**)&D_p_img_sub_4d, sz_img_sub_read[0] * sz_img_sub_read[1] * sz_img_sub_read[2] * sz_img_sub_read[3] * sizeof(unsigned char));
	//hipMalloc((void**)&D_p_img_warp_4d, sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(unsigned char));
	//hipMalloc((void**)&D_sz_img_sub, 4 * sizeof(float));

	//H_sz_img_sub[0] = sz_img_sub[0]; H_sz_img_sub[1] = sz_img_sub[1]; H_sz_img_sub[2] = sz_img_sub[2]; H_sz_img_sub[3] = sz_img_sub[3];

	//for (V3DLONG a = 0; a < sz_img_sub[2]; a++)
	//{
	//	for (V3DLONG b = 0; b < sz_img_sub[1]; b++)
	//	{
	//		for (V3DLONG c = 0; c < sz_img_sub[0]; c++)
	//		{
	//			printf("\tTime to generate: %3.1f \n", p_img_sub_4d[0][a][b][c]);
	//		}
	//	}
	//}
	//for (V3DLONG a = 0; a < sz_img_sub[2]; a++)
	//{
	//	for (V3DLONG b = 0; b < sz_img_sub[1]; b++)
	//	{
	//		for (V3DLONG c = 0; c < sz_img_sub[0]; c++)
	//		{
	//			H_p_img_sub_4d[a*sz_img_sub[1] * sz_img_sub[0] + b*sz_img_sub[0]+c] = p_img_sub_4d[0][a][b][c];
	//		}
	//	}
	//}
	int aa = sz_img_sub[0]; int bb = sz_img_sub[1]; int cc = sz_img_sub[2]; int dd = sz_img_sub[3];
	//int aa_read = sz_img_sub_read[0]; int bb_read = sz_img_sub_read[1]; int cc_read = sz_img_sub_read[2]; int dd_read = sz_img_sub_read[3];
	for (V3DLONG a = 0; a < gsz2; a++)
	{
		for (V3DLONG b = 0; b < gsz1; b++)
		{
			for (V3DLONG c = 0; c < gsz0; c++)
			{

				H_pppSubDF_x[a*gsz1*gsz0 + b*gsz0 + c] = pppSubDF[a][b][c].sx;
				H_pppSubDF_y[a*gsz1*gsz0 + b*gsz0 + c] = pppSubDF[a][b][c].sy;
				H_pppSubDF_z[a*gsz1*gsz0 + b*gsz0 + c] = pppSubDF[a][b][c].sz;

			}
		}
	}

	for (int i = 0; i < x_bsplinebasis.nrows(); i++)
	{
		for (int j = 0; j < x_bsplinebasis.ncols(); j++)
		{

			H_x_bsplinebasis[i * x_bsplinebasis.ncols() + j] = x_bsplinebasis(i + 1, j + 1);

		}
	}


	HANDLE_ERROR(hipMemcpy(D_pppSubDF_x, H_pppSubDF_x, gsz2 * gsz1 *gsz0 * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppSubDF_y, H_pppSubDF_y, gsz2 * gsz1 *gsz0 * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppSubDF_z, H_pppSubDF_z, gsz2 * gsz1 *gsz0 * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_x_bsplinebasis, H_x_bsplinebasis, x_bsplinebasis.nrows() * x_bsplinebasis.ncols() * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppDFBlock_x, H_pppDFBlock_x, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppDFBlock_y, H_pppDFBlock_y, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppDFBlock_z, H_pppDFBlock_z, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float), hipMemcpyHostToDevice));

	//HANDLE_ERROR(hipMemcpy(D_sz_img_sub, H_sz_img_sub, 4 * sizeof(float), hipMemcpyHostToDevice)); printf("mark8");
	//HANDLE_ERROR(hipMemcpy(D_p_img_sub_4d, H_p_img_sub_4d, 2 * sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float), hipMemcpyHostToDevice));
	//HANDLE_ERROR(hipMemcpy(D_p_img_sub_4d, p_img_sub, sz_img_sub_read[0] * sz_img_sub_read[1] * sz_img_sub_read[2] * sz_img_sub_read[3] * sizeof(unsigned char), hipMemcpyHostToDevice));
	//HANDLE_ERROR(hipMemcpy(D_p_img_warp_4d, H_p_img_warp_4d, sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float), hipMemcpyHostToDevice));
	//	printf("\tsz_img_sub[0]:%d sz_img_sub[1]:%d sz_img_sub[2]:%d sz_img_sub[3]:%d\n", sz_img_sub[0], sz_img_sub[1], sz_img_sub[2], sz_img_sub[3]);




	for (int k = 0; k < gsz2 - 1 - 2; k++)
	{
		dim3 grid((gsz0 - 1 - 2 + threads_num - 1) / threads_num, (gsz1 - 1 - 2 + threads_num - 1) / threads_num);
		dim3 block(threads_num, threads_num);
		get_Displacement_brightness_sort << <grid, block >> >(sz_gridwnd, k, gsz1, gsz0, D_pppSubDF_x, D_pppSubDF_y, D_pppSubDF_z, D_x_bsplinebasis, D_pppDFBlock_x, D_pppDFBlock_y,
			D_pppDFBlock_z, szBlock_x, szBlock_y, szBlock_z, aa, bb, cc, dd, i_interpmethod_img, start_block_x, start_block_y, start_block_z, gs_ori2, gs_ori1, gs_ori0, D_sort_x, D_sort_y, D_sort_z);
		//get_Displacement << <grid, block >> >(sz_gridwnd, k, gsz1, gsz0, D_pppSubDF_x, D_pppSubDF_y, D_pppSubDF_z, D_x_bsplinebasis, D_pppDFBlock_x, D_pppDFBlock_y, D_pppDFBlock_z);
		//get_brightness << <grid, block >> >(gsz1, gsz0, D_p_img_sub_4d, D_sz_img_sub, D_pppDFBlock_x, D_pppDFBlock_y, D_pppDFBlock_z, szBlock_x, szBlock_y, szBlock_z, i_interpmethod_img, k, D_p_img_warp_4d);
		hipDeviceSynchronize();
	}



	//printf("mark1\n");
	//HANDLE_ERROR(hipMemcpy(p_img_warp, D_p_img_warp_4d, sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(unsigned char), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(sort_x, D_sort_x, sz_img_sub[0] * sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(sort_y, D_sort_y, sz_img_sub[1] * sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(sort_z, D_sort_z, sz_img_sub[2] * sizeof(float), hipMemcpyDeviceToHost));

	//printf("mark2\n");






	//for (V3DLONG a = 0; a < sz_img_sub[2]; a++)
	//{
	//	for (V3DLONG b = 0; b < sz_img_sub[1]; b++)
	//	{
	//		for (V3DLONG c = 0; c < sz_img_sub[0]; c++)
	//		{
	//			p_img_warp_4d[0][a][b][c] = H_p_img_warp_4d[a*sz_img_sub[1] * sz_img_sub[0] + b*sz_img_sub[0] + c];
	//			//if (H_p_img_warp_4d[a*sz_img_sub[1] * sz_img_sub[0] + b*sz_img_sub[0] + c]!=0)printf("p_img_warp:%f", H_p_img_warp_4d[a*sz_img_sub[1] * sz_img_sub[0] + b*sz_img_sub[0] + c]);
	//		}
	//	}
	//}





	//HANDLE_ERROR(hipEventRecord(stop, 0));
	//HANDLE_ERROR(hipEventSynchronize(stop));
	//float elapsedTime;
	//HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	//printf("\tTime to generate: %3.1f ms\n", elapsedTime);
	//HANDLE_ERROR(hipEventDestroy(start));
	//HANDLE_ERROR(hipEventDestroy(stop));
	free(H_pppSubDF_x);
	free(H_pppSubDF_y);
	free(H_pppSubDF_z);

	free(H_x_bsplinebasis);

	free(H_pppDFBlock_x);
	free(H_pppDFBlock_y);
	free(H_pppDFBlock_z);

	hipFree(D_pppSubDF_x); hipFree(D_pppSubDF_y); hipFree(D_pppSubDF_z);
	hipFree(D_pppDFBlock_x); hipFree(D_pppDFBlock_y); hipFree(D_pppDFBlock_z);
	hipFree(D_x_bsplinebasis); hipFree(D_p_img_sub_4d); hipFree(D_p_img_warp_4d);
	hipFree(D_sort_x); hipFree(D_sort_y); hipFree(D_sort_z);
	return true;
}



extern "C" bool gpu_interpolation_stps(const int gsz2, const int gsz1, const int gsz0, DisplaceFieldF3D ***&pppSubDF, const Matrix &x_bsplinebasis, const V3DLONG sz_gridwnd, DisplaceFieldF3D ***&pppDFBlock,
	unsigned char ****&p_img_sub_4d, const V3DLONG *sz_img_sub, const V3DLONG szBlock_x, const V3DLONG szBlock_y, const V3DLONG szBlock_z, const int i_interpmethod_img, unsigned char ****&p_img_warp_4d,
	const V3DLONG *sz_img_sub_ori, const unsigned char *p_img_sub, unsigned char *p_img_warp)
{
	//hipEvent_t   start, stop;
	//HANDLE_ERROR(hipEventCreate(&start));
	//HANDLE_ERROR(hipEventCreate(&stop));
	float *H_pppSubDF_x, *H_pppSubDF_y, *H_pppSubDF_z, *H_pppDFBlock_x, *H_pppDFBlock_y, *H_pppDFBlock_z;
	float *H_x_bsplinebasis, *D_x_bsplinebasis;
	float *D_pppSubDF_x, *D_pppSubDF_y, *D_pppSubDF_z, *D_pppDFBlock_x, *D_pppDFBlock_y, *D_pppDFBlock_z;
	H_pppSubDF_x = (float*)malloc(gsz2 * gsz1 *gsz0 * sizeof(float));
	H_pppSubDF_y = (float*)malloc(gsz2 * gsz1 *gsz0 * sizeof(float));
	H_pppSubDF_z = (float*)malloc(gsz2 * gsz1 *gsz0 * sizeof(float));

	H_x_bsplinebasis = (float*)malloc(x_bsplinebasis.nrows() * x_bsplinebasis.ncols() * sizeof(float));
	H_pppDFBlock_x = (float*)malloc(sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));
	H_pppDFBlock_y = (float*)malloc(sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));
	H_pppDFBlock_z = (float*)malloc(sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));
	//printf("mark2\n");
	hipMalloc((void**)&D_pppSubDF_x, gsz2 * gsz1 *gsz0 * sizeof(float));
	hipMalloc((void**)&D_pppSubDF_y, gsz2 * gsz1 *gsz0 * sizeof(float));
	hipMalloc((void**)&D_pppSubDF_z, gsz2 * gsz1 *gsz0 * sizeof(float));
	hipMalloc((void**)&D_x_bsplinebasis, x_bsplinebasis.nrows() * x_bsplinebasis.ncols() * sizeof(float));
	hipMalloc((void**)&D_pppDFBlock_x, sz_gridwnd * sz_gridwnd *sz_gridwnd* sizeof(float));
	hipMalloc((void**)&D_pppDFBlock_y, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));
	hipMalloc((void**)&D_pppDFBlock_z, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float));


	unsigned char *D_p_img_sub_4d, *D_p_img_warp_4d;
	//H_p_img_sub_4d = (float*)malloc(2 * sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float));
	//H_p_img_sub_4d = (float*)malloc(sz_img_sub_ori[0] * sz_img_sub_ori[1] * sz_img_sub_ori[2] * sz_img_sub[3] * sizeof(float));
	//H_p_img_warp_4d = (float*)malloc(sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float));
	//H_sz_img_sub = (float*)malloc(4 * sizeof(float));
	//hipMalloc((void**)&D_p_img_sub_4d, 2 * sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float));
	hipMalloc((void**)&D_p_img_sub_4d, sz_img_sub_ori[0] * sz_img_sub_ori[1] * sz_img_sub_ori[2] * sz_img_sub[3] * sizeof(unsigned char));
	hipMalloc((void**)&D_p_img_warp_4d, sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(unsigned char));
	//hipMalloc((void**)&D_sz_img_sub, 4 * sizeof(float));

	//H_sz_img_sub[0] = sz_img_sub[0]; H_sz_img_sub[1] = sz_img_sub[1]; H_sz_img_sub[2] = sz_img_sub[2]; H_sz_img_sub[3] = sz_img_sub[3];

	//for (V3DLONG a = 0; a < sz_img_sub[2]; a++)
	//{
	//	for (V3DLONG b = 0; b < sz_img_sub[1]; b++)
	//	{
	//		for (V3DLONG c = 0; c < sz_img_sub[0]; c++)
	//		{
	//			printf("\tTime to generate: %3.1f \n", p_img_sub_4d[0][a][b][c]);
	//		}
	//	}
	//}
	//for (V3DLONG a = 0; a < sz_img_sub[2]; a++)
	//{
	//	for (V3DLONG b = 0; b < sz_img_sub[1]; b++)
	//	{
	//		for (V3DLONG c = 0; c < sz_img_sub[0]; c++)
	//		{
	//			H_p_img_sub_4d[a*sz_img_sub[1] * sz_img_sub[0] + b*sz_img_sub[0]+c] = p_img_sub_4d[0][a][b][c];
	//		}
	//	}
	//}
	int aa = sz_img_sub[0]; int bb = sz_img_sub[1]; int cc = sz_img_sub[2]; int dd = sz_img_sub[3];
	int aa_ori = sz_img_sub_ori[0]; int bb_ori = sz_img_sub_ori[1]; int cc_ori = sz_img_sub_ori[2]; int dd_ori = sz_img_sub_ori[3];
	for (V3DLONG a = 0; a < gsz2; a++)
	{
		for (V3DLONG b = 0; b < gsz1; b++)
		{
			for (V3DLONG c = 0; c < gsz0; c++)
			{

				H_pppSubDF_x[a*gsz1*gsz0 + b*gsz0 + c] = pppSubDF[a][b][c].sx;
				H_pppSubDF_y[a*gsz1*gsz0 + b*gsz0 + c] = pppSubDF[a][b][c].sy;
				H_pppSubDF_z[a*gsz1*gsz0 + b*gsz0 + c] = pppSubDF[a][b][c].sz;

			}
		}
	}

	for (int i = 0; i < x_bsplinebasis.nrows(); i++)
	{
		for (int j = 0; j < x_bsplinebasis.ncols(); j++)
		{

			H_x_bsplinebasis[i * x_bsplinebasis.ncols() + j] = x_bsplinebasis(i + 1, j + 1);

		}
	}

	/*for (V3DLONG a = 0; a < sz_img_sub_ori[2]; a++)
	{
	for (V3DLONG b = 0; b < sz_img_sub_ori[1]; b++)
	{
	for (V3DLONG c = 0; c < sz_img_sub_ori[0]; c++)
	{
	H_p_img_sub_4d[a * sz_img_sub_ori[1] * sz_img_sub_ori[0] + b * sz_img_sub_ori[0] + c] = p_img_sub_4d[0][a][b][c];
	}
	}
	}*/
	//for (int i = 0; i < x_bsplinebasis.nrows(); i++)
	//{
	//	printf("\n��%d��", i);
	//	for (int j = 0; j < x_bsplinebasis.ncols(); j++)
	//	{
	//		printf("%.3f\t",H_x_bsplinebasis[i * x_bsplinebasis.ncols() + j]);
	//	}
	//	printf("\n");
	//}
	HANDLE_ERROR(hipMemcpy(D_pppSubDF_x, H_pppSubDF_x, gsz2 * gsz1 *gsz0 * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppSubDF_y, H_pppSubDF_y, gsz2 * gsz1 *gsz0 * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppSubDF_z, H_pppSubDF_z, gsz2 * gsz1 *gsz0 * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_x_bsplinebasis, H_x_bsplinebasis, x_bsplinebasis.nrows() * x_bsplinebasis.ncols() * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppDFBlock_x, H_pppDFBlock_x, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppDFBlock_y, H_pppDFBlock_y, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_pppDFBlock_z, H_pppDFBlock_z, sz_gridwnd * sz_gridwnd *sz_gridwnd * sizeof(float), hipMemcpyHostToDevice));

	//HANDLE_ERROR(hipMemcpy(D_sz_img_sub, H_sz_img_sub, 4 * sizeof(float), hipMemcpyHostToDevice)); printf("mark8");
	//HANDLE_ERROR(hipMemcpy(D_p_img_sub_4d, H_p_img_sub_4d, 2 * sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_p_img_sub_4d, p_img_sub, sz_img_sub_ori[0] * sz_img_sub_ori[1] * sz_img_sub_ori[2] * sz_img_sub[3] * sizeof(unsigned char), hipMemcpyHostToDevice));
	//HANDLE_ERROR(hipMemcpy(D_p_img_warp_4d, H_p_img_warp_4d, sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(float), hipMemcpyHostToDevice));
	//	printf("\tsz_img_sub[0]:%d sz_img_sub[1]:%d sz_img_sub[2]:%d sz_img_sub[3]:%d\n", sz_img_sub[0], sz_img_sub[1], sz_img_sub[2], sz_img_sub[3]);


	if (i_interpmethod_img)//nearest neighbor interpolate
	{
		for (int k = 0; k < gsz2 - 1 - 2; k++)
		{
			dim3 grid((gsz0 - 1 - 2 + threads_num - 1) / threads_num, (gsz1 - 1 - 2 + threads_num - 1) / threads_num);
			dim3 block(threads_num, threads_num);
			get_Displacement_brightness_nn << <grid, block >> >(sz_gridwnd, k, gsz1, gsz0, D_pppSubDF_x, D_pppSubDF_y, D_pppSubDF_z, D_x_bsplinebasis, D_pppDFBlock_x, D_pppDFBlock_y, D_pppDFBlock_z, szBlock_x, szBlock_y, szBlock_z, aa, bb, cc, dd, D_p_img_sub_4d, D_p_img_warp_4d, i_interpmethod_img, aa_ori, bb_ori, cc_ori, dd_ori);
			//get_Displacement << <grid, block >> >(sz_gridwnd, k, gsz1, gsz0, D_pppSubDF_x, D_pppSubDF_y, D_pppSubDF_z, D_x_bsplinebasis, D_pppDFBlock_x, D_pppDFBlock_y, D_pppDFBlock_z);
			//get_brightness << <grid, block >> >(gsz1, gsz0, D_p_img_sub_4d, D_sz_img_sub, D_pppDFBlock_x, D_pppDFBlock_y, D_pppDFBlock_z, szBlock_x, szBlock_y, szBlock_z, i_interpmethod_img, k, D_p_img_warp_4d);
			hipDeviceSynchronize();
		}
	}
	else//linear interpolate
	{
		for (int k = 0; k < gsz2 - 1 - 2; k++)
		{
			dim3 grid((gsz0 - 1 - 2 + threads_num - 1) / threads_num, (gsz1 - 1 - 2 + threads_num - 1) / threads_num);
			dim3 block(threads_num, threads_num);
			get_Displacement_brightness_stps << <grid, block >> >(sz_gridwnd, k, gsz1, gsz0, D_pppSubDF_x, D_pppSubDF_y, D_pppSubDF_z, D_x_bsplinebasis, D_pppDFBlock_x, D_pppDFBlock_y, D_pppDFBlock_z, szBlock_x, szBlock_y, szBlock_z, aa, bb, cc, dd, D_p_img_sub_4d, D_p_img_warp_4d, i_interpmethod_img, aa_ori, bb_ori, cc_ori, dd_ori);
			//get_Displacement << <grid, block >> >(sz_gridwnd, k, gsz1, gsz0, D_pppSubDF_x, D_pppSubDF_y, D_pppSubDF_z, D_x_bsplinebasis, D_pppDFBlock_x, D_pppDFBlock_y, D_pppDFBlock_z);
			//get_brightness << <grid, block >> >(gsz1, gsz0, D_p_img_sub_4d, D_sz_img_sub, D_pppDFBlock_x, D_pppDFBlock_y, D_pppDFBlock_z, szBlock_x, szBlock_y, szBlock_z, i_interpmethod_img, k, D_p_img_warp_4d);
			hipDeviceSynchronize();
		}
	}




	//printf("mark3\n");
	//printf("mark1\n");
	HANDLE_ERROR(hipMemcpy(p_img_warp, D_p_img_warp_4d, sz_img_sub[0] * sz_img_sub[1] * sz_img_sub[2] * sz_img_sub[3] * sizeof(unsigned char), hipMemcpyDeviceToHost));
	//printf("mark2\n");






	//for (V3DLONG a = 0; a < sz_img_sub[2]; a++)
	//{
	//	for (V3DLONG b = 0; b < sz_img_sub[1]; b++)
	//	{
	//		for (V3DLONG c = 0; c < sz_img_sub[0]; c++)
	//		{
	//			p_img_warp_4d[0][a][b][c] = H_p_img_warp_4d[a*sz_img_sub[1] * sz_img_sub[0] + b*sz_img_sub[0] + c];
	//			//if (H_p_img_warp_4d[a*sz_img_sub[1] * sz_img_sub[0] + b*sz_img_sub[0] + c]!=0)printf("p_img_warp:%f", H_p_img_warp_4d[a*sz_img_sub[1] * sz_img_sub[0] + b*sz_img_sub[0] + c]);
	//		}
	//	}
	//}





	//HANDLE_ERROR(hipEventRecord(stop, 0));
	//HANDLE_ERROR(hipEventSynchronize(stop));
	//float elapsedTime;
	//HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	//printf("\tTime to generate: %3.1f ms\n", elapsedTime);
	//HANDLE_ERROR(hipEventDestroy(start));
	//HANDLE_ERROR(hipEventDestroy(stop));
	free(H_pppSubDF_x);
	free(H_pppSubDF_y);
	free(H_pppSubDF_z);

	free(H_x_bsplinebasis);

	free(H_pppDFBlock_x);
	free(H_pppDFBlock_y);
	free(H_pppDFBlock_z);

	hipFree(D_pppSubDF_x); hipFree(D_pppSubDF_y); hipFree(D_pppSubDF_z);
	hipFree(D_pppDFBlock_x); hipFree(D_pppDFBlock_y); hipFree(D_pppDFBlock_z);
	hipFree(D_x_bsplinebasis); hipFree(D_p_img_sub_4d); hipFree(D_p_img_warp_4d);
	return true;
}