#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsolver.h>
#include "ele.cuh"
#include"newmat.h"

__global__ void get_gpu_extendornormal(float *d_Q, int ncpt, int i, int k, float *X, float *SSR)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= ncpt)return;

	float sum = 0;
	for (int j = 0; j < i; j++)
	{
		sum += -d_Q[idx*ncpt + j] * d_Q[(k - 1)*ncpt + j];
	}

	X[idx] = sum;
	////	printf("\nֵΪ��%f\t", sum);
	//	__syncthreads();
	//	if (idx == (k-1))
	//	{
	//		X[idx] += 1.0;
	//		float SumSquare = 0;
	//		for (int j = 0; j < ncpt; j++)
	//		{
	//			SumSquare += X[j] * X[j];
	//		}
	//		for (int j = 0; j < ncpt; j++)
	//		{
	//			X[j] /= sqrt(SumSquare);
	//		}
	//		
	//	}
	//	__syncthreads();
	//	d_Q[idx*ncpt + i ] = X[idx];
	//	SSR[idx] += X[idx] * X[idx];
	//	//__syncthreads();
}
__global__ void get_gpu_assign(float *d_Q, int ncpt, int i, float *X, float *SSR)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= ncpt)return;


	d_Q[idx*ncpt + i] = X[idx];
	SSR[idx] += X[idx] * X[idx];
	//__syncthreads();
}

extern "C" int gpu_extendornormal(int ncpt, int n, Matrix &Q)
{
	ColumnVector SSR;
	//ColumnVector X;
	//Matrix A1 = Q.Columns(1, n);
	SSR = Q.sum_square_rows();//��ֵÿ��Ԫ�ص�ƽ���ͣ�Ԫ�ش�1��ʼ����
	//printf("\t>>t time consume %f\n", SSR(1)); printf("\t>>t time consume %f\n", SSR(2)); printf("\t>>t time consume %f\n", SSR(3));
	//int t; SSR.minimum1(t); printf("\t>>t time consume %d\n", t);
	float *h_Q, *d_Q, *h_SSR, *d_SSR, *h_X, *d_X;
	h_Q = (float*)malloc(ncpt*ncpt * sizeof(float));  hipMalloc((void**)&d_Q, ncpt*ncpt * sizeof(float));
	h_SSR = (float*)malloc(ncpt * sizeof(float));  hipMalloc((void**)&d_SSR, ncpt * sizeof(float));
	h_X = (float*)malloc(ncpt * sizeof(float));  hipMalloc((void**)&d_X, ncpt * sizeof(float));
	for (int i = 0; i < Q.nrows(); i++)
	{
		for (int j = 0; j < Q.ncols(); j++)
		{

			h_Q[i*ncpt + j] = Q(i + 1, j + 1);

		}
	}
	for (int j = 1; j <= ncpt; j++)
	{

		h_SSR[j - 1] = SSR(j);

	}
	HANDLE_ERROR(hipMemcpy(d_Q, h_Q, ncpt* ncpt * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_SSR, h_SSR, ncpt * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_X, h_X, ncpt * sizeof(float), hipMemcpyHostToDevice));
	for (int i = n; i < ncpt; ++i)
	{

		int k; SSR.minimum1(k);// printf("\nkֵΪ��%d\t", k);//������СԪ�ص��к�
		//// orthogonalise column with 1 at element k, 0 elsewhere
		////printf("\t>> %d\n", i);// next line is rather inefficient
		//ColumnVector X;
		int block = 64;
		int grid = (ncpt + 64 - 1) / 64;
		get_gpu_extendornormal << <grid, block >> >(d_Q, ncpt, i, k, d_X, d_SSR);
		HANDLE_ERROR(hipMemcpy(h_X, d_X, ncpt * sizeof(float), hipMemcpyDeviceToHost));
		h_X[k - 1] += 1.0;
		float SumSquare = 0;
		for (int j = 0; j < ncpt; j++)
		{
			SumSquare += h_X[j] * h_X[j];
		}
		for (int j = 0; j < ncpt; j++)
		{
			h_X[j] /= sqrt(SumSquare);
		}
		HANDLE_ERROR(hipMemcpy(d_X, h_X, ncpt * sizeof(float), hipMemcpyHostToDevice));
		get_gpu_assign << <grid, block >> >(d_Q, ncpt, i, d_X, d_SSR);
		//printf("\nkֵΪ��%d\t", k);
		//get_gpu_extendornormal << <1, 3 >> >(d_Q, ncpt, i, k, d_X, d_SSR);
		HANDLE_ERROR(hipMemcpy(h_SSR, d_SSR, ncpt * sizeof(float), hipMemcpyDeviceToHost));
		for (int j = 1; j <= ncpt; j++)
		{

			SSR(j) = h_SSR[j - 1];

		}

		////ColumnVector X = -Q.Columns(1, i) * Q.SubMatrix(k, k, 1, i).t();
		//X(k) += 1.0;
		//// normalise
		//X /= sqrt(X.SumSquare());	//for (k = 1; k <= nr; ++k) printf("\t>> %f\n", X(k));
		//// update row sums of squares
		//for (k = 1; k <= ncpt; ++k) SSR(k) += X(k)*X(k);
		//// load new column into matrix
		//Q.Column(i + 1) = X;

	}
	HANDLE_ERROR(hipMemcpy(h_Q, d_Q, ncpt*ncpt * sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(h_X, d_X, ncpt * sizeof(float), hipMemcpyDeviceToHost));
	//for (int i = 0; i < ncpt*ncpt; i++)printf("\nֵΪ��%f\t", h_Q[i]);



	for (int i = 0; i < Q.nrows(); i++)
	{
		for (int j = 0; j < Q.ncols(); j++)
		{

			Q(i + 1, j + 1) = h_Q[i*ncpt + j];

		}
	}
	free(h_Q);
	free(h_SSR);
	free(h_X);
	hipFree(&d_Q); hipFree(&d_X);
	hipFree(&d_SSR);
	HANDLE_ERROR(hipDeviceReset());
	return 0;

}