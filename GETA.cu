#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <malloc.h>
#include<time.h>
#include"newmat.h"
#include "ele.cuh"

//__global__ void matrixInv(float* A, float* B, int i, int wA)
//{
//	int bx = blockIdx.x;
//	int by = blockIdx.y;
//	int tx = threadIdx.x;
//	int ty = threadIdx.y;
//
//	const int index_tx = bx*blockDim.x + tx;
//	const int index_ty = by*blockDim.y + ty;
//	/*int index=index_ty+(index_tx)*wA;*/
//	const float temp = A[i*wA + i];
//	if (index_ty < wA&&index_tx < wA)
//	{
//		B[i*wA + index_tx] = B[i*wA + index_tx] / temp;
//		A[i*wA + index_tx] = A[i*wA + index_tx] / temp;
//	}
//	float Avalue = 0.0;
//	float Bvalue = 0.0;
//	// C[index_ty*wA+index_tx]=A[index_ty*wA+i]*A[i*n+index_tx];
//	// D[index_ty*wA+index_tx]=A[index_ty*wA+i]*B[i*n+index_tx];
//
//	__shared__ float As[BLOCK_DIM][BLOCK_DIM];
//	__shared__ float Bs[BLOCK_DIM][BLOCK_DIM];
//	__shared__ float Cs[BLOCK_DIM][BLOCK_DIM];
//	__shared__ float Ds[BLOCK_DIM][BLOCK_DIM];
//	__shared__ float Ms[BLOCK_DIM][BLOCK_DIM];
//	if (index_ty < wA&&index_tx < wA)
//		for (int m = 0; m < wA / TILE_WIDTH; ++m)
//		{
//			Bs[ty][tx] = B[index_ty*wA + (m*TILE_WIDTH + tx)];
//			As[ty][tx] = A[index_ty*wA + (m*TILE_WIDTH + tx)];
//			Cs[ty][tx] = A[(m*TILE_WIDTH + ty)*wA + i];
//			Ds[ty][tx] = B[i*wA + (m*TILE_WIDTH + tx)];
//			Ms[ty][tx] = A[i*wA + (m*TILE_WIDTH + tx)];
//			__syncthreads();
//			/* int k=i-(i/TILE_WIDTH)*TILE_WIDTH;*/
//
//			// const int k=i%wA;
//			// const int j=i/wA;  
//
//			Avalue = A[index_ty*wA + i] * A[i*wA + index_tx];
//			Bvalue = A[index_ty*wA + i] * B[i*wA + index_tx];
//
//			Bs[ty][tx] = Bs[ty][tx] - Bvalue;
//			As[ty][tx] = As[ty][tx] - Avalue;
//			__syncthreads();
//			if (index_ty != i)
//			{
//				B[index_ty*wA + index_tx] = Bs[ty][tx];
//				A[index_ty*wA + index_tx] = As[ty][tx];
//			}
//		}
//}
__global__ void get_bTWOKERNAL(const Matrixtran dev_q2, const Matrixtran dev_xnxn_K, Matrixtran  dev_A1)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col >= dev_A1.width || row >= dev_A1.height)
		return;
	float sum = 0;
	for (int i = 0; i < dev_q2.height; i++)
	{
		sum += dev_q2.elements[i*dev_q2.width + row] * dev_xnxn_K.elements[i*dev_xnxn_K.width + col];
	}
	dev_A1.elements[dev_A1.width*row + col] = sum;
}
__global__ void get_g(const Matrixtran dev_A, const Matrixtran dev_q2, Matrixtran  dev_Af)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col >= dev_Af.width || row >= dev_Af.height)
		return;
	float sum = 0;
	for (int i = 0; i < dev_A.width; i++)
	{
		sum += dev_A.elements[row*dev_A.width + i] * dev_q2.elements[i*dev_q2.width + col];
	}
	dev_Af.elements[dev_Af.width*row + col] = sum;


}
extern "C" bool gpu_A(int ncpt, const Matrix &q2_t, const Matrix &xnxn_K, Matrix &A)
{

	hipEvent_t   start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));
	printf("\t---------------------cuda process start---------------------\n");
	printf("\tncpt:%d\n", ncpt);
	//�����ڴ�
	Matrixtran dev_q2_t, dev_xnxn_K, host_q2_t, host_xnxn_K, host_A1, dev_A1, host_A, dev_A;
	host_q2_t.width = ncpt - 4; host_q2_t.height = ncpt; size_t size_host_q2_t = host_q2_t.width*host_q2_t.height*sizeof(float); host_q2_t.elements = (float*)malloc(host_q2_t.height*host_q2_t.width * sizeof(float));
	host_xnxn_K.width = ncpt; host_xnxn_K.height = ncpt; size_t size_host_xnxn_K = host_xnxn_K.width*host_xnxn_K.height*sizeof(float); host_xnxn_K.elements = (float*)malloc(host_xnxn_K.height*host_xnxn_K.width * sizeof(float));;
	dev_q2_t.width = ncpt - 4; dev_q2_t.height = ncpt; size_t size_dev_q2_t = dev_q2_t.width*dev_q2_t.height*sizeof(float); hipMalloc((void**)&dev_q2_t.elements, size_dev_q2_t);
	dev_xnxn_K.width = ncpt; dev_xnxn_K.height = ncpt; size_t size_dev_xnxn_K = dev_xnxn_K.width*dev_xnxn_K.height*sizeof(float); hipMalloc((void**)&dev_xnxn_K.elements, size_dev_xnxn_K);
	host_A1.width = ncpt; host_A1.height = ncpt - 4; size_t size_host_A1 = host_A1.width*host_A1.height*sizeof(float); host_A1.elements = (float*)malloc(host_A1.height*host_A1.width * sizeof(float));
	dev_A1.width = ncpt; dev_A1.height = ncpt - 4; size_t size_dev_A1 = dev_A1.width*dev_A1.height*sizeof(float); hipMalloc((void**)&dev_A1.elements, size_dev_A1);
	host_A.width = ncpt - 4; host_A.height = ncpt - 4; size_t size_host_A = host_A.width*host_A.height*sizeof(float); host_A.elements = (float*)malloc(host_A.height*host_A.width * sizeof(float));
	dev_A.width = ncpt - 4; dev_A.height = ncpt - 4; size_t size_dev_A = dev_A.width*dev_A.height*sizeof(float); hipMalloc((void**)&dev_A.elements, size_dev_A);


	//ת����������
	for (int i = 0; i < q2_t.nrows(); i++)
	{
		for (int j = 0; j < q2_t.ncols(); j++)
		{

			host_q2_t.elements[i*host_q2_t.width + j] = q2_t(i + 1, j + 1);

		}
	}

	for (int i = 0; i < xnxn_K.nrows(); i++)
	{
		for (int j = 0; j < xnxn_K.ncols(); j++)
		{
			host_xnxn_K.elements[i*host_xnxn_K.width + j] = xnxn_K(i + 1, j + 1);

		}
	}


	//��cpu�����ڴ浽gpu
	HANDLE_ERROR(hipMemcpy(dev_q2_t.elements, host_q2_t.elements, dev_q2_t.width* dev_q2_t.height * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_xnxn_K.elements, host_xnxn_K.elements, host_xnxn_K.width* host_xnxn_K.height * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_A1.elements, host_A1.elements, host_A1.width* host_A1.height * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_A.elements, host_A.elements, host_A.width* host_A.height * sizeof(float), hipMemcpyHostToDevice));



	dim3 devgrid((ncpt - 4 + threads_num - 1) / threads_num, (ncpt + threads_num - 1) / threads_num);
	dim3 devblock(threads_num, threads_num);

	//get_b << <devgrid, devblock >> >(dev_q2_t, dev_xnxn_K, dev_A1, dev_A);
	get_bTWOKERNAL << <devgrid, devblock >> >(dev_q2_t, dev_xnxn_K, dev_A1);
	get_g << <devgrid, devblock >> >(dev_A1, dev_q2_t, dev_A);
	//��gpu�����ڴ浽cpu
	//HANDLE_ERROR(hipMemcpy(host_A1.elements, dev_A1.elements, dev_A1.width* dev_A1.height * sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(host_A.elements, dev_A.elements, dev_A.width* dev_A.height * sizeof(float), hipMemcpyDeviceToHost));
	//printf("\n");
	//for (int j = 0; j < size_dev_A1; j++)
	//{
	//	
	//	printf("%.3f\t", host_A1.elements[j]);
	//	if (((j + 1) % host_A1.width) == 0){ printf("\n��%d��", (j + 1) / host_A1.width); }
	//
	//}
	//ת������
	for (int i = 0; i < A.nrows(); i++)
	{
		for (int j = 0; j < A.ncols(); j++)
		{
			A(i + 1, j + 1) = host_A.elements[i*host_A.width + j];

		}
	}
	//for (long long row = 1; row <= A.nrows(); row++)
	//{
	//	printf("\n��%d��", row);
	//	for (long long col = 1; col <= A.ncols(); col++)
	//		printf("%.3f\t", A(row, col));
	//	printf("\n");
	//}
	//��ʱ����
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("\tTime to generate: %3.1f ms\n", elapsedTime);
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	printf("\t---------------------cuda process end---------------------\n");
	//�ͷ��ڴ�
	hipFree(&dev_q2_t);
	hipFree(&dev_xnxn_K);
	hipFree(&dev_A1);
	hipFree(&dev_A);
	HANDLE_ERROR(hipDeviceReset());
	return true;
}
