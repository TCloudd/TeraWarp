#include <stdio.h>
#include <stdlib.h>
#include <hipsolver.h>
#include "ele.cuh"
#include"newmat.h"


extern "C" int gpu_QR(int ncpt, const Matrix &A, Matrix &Q, Matrix &R)
{

	hipsolverHandle_t cusolverH = NULL;
	const int m = ncpt;
	const int n = ncpt;
	const int lda = ncpt;
	float *q;
	q = (float*)malloc(m*n * sizeof(float));
	float *r;
	r = (float*)malloc(m*n * sizeof(float));

	for (int i = 0; i < ncpt; i++)
	{
		for (int j = 0; j < ncpt; j++)
		{

			q[j*ncpt + i] = A(i + 1, j + 1);

		}
	}

	float *W;
	W = (float*)malloc(m * sizeof(float));
	int info_gpu = 0;//����״̬����
	// ����1���������
	hipsolverDnCreate(&cusolverH);
	// ����2�������Դ�ռ�
	float *d_A = NULL; hipMalloc((void**)&d_A, sizeof(float) * lda * m);//����Hermite�������������������Ϊͬһ�ռ䣩
	float *d_W = NULL; hipMalloc((void**)&d_W, sizeof(float) *m);//��������ֵ�洢�ռ�
	int *devInfo = NULL; hipMalloc((void**)&devInfo, sizeof(int));//����������״̬�ռ�
	hipMemcpy(d_A, q, sizeof(float) * lda * m, hipMemcpyHostToDevice);//���ݿ���
	hipMemcpy(d_W, W, sizeof(float)* m, hipMemcpyHostToDevice);//���ݿ���
	// ����3��������㻺��ռ䣬�����Դ�������ÿռ�
	float *d_work = NULL; float *h_work = NULL;
	int lwork = 0;
	int hwork = 0;
	//hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
	//hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipsolverDnSgeqrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork);
	hipMalloc((void**)&d_work, sizeof(float)*lwork);
	hipsolverDnSgeqrf(cusolverH, m, m, d_A, lda, d_W, d_work, lwork, devInfo);
	hipDeviceSynchronize();
	hipMemcpy(q, d_A, sizeof(float)*lda*m, hipMemcpyDeviceToHost);
	float *B = (float*)malloc(lda*m * sizeof(float));
	/*	for (int j = 0; j < m*m; j++)
	{
	if (((j) % m) == 0){ printf("\n��%d��", (((j + 1) / m) + 1)); }
	printf("%.3f\t", q[j]);
	}*/
	for (long long row = 0; row < m; row++)
	{

		for (long long col = row; col < m; col++)
			B[row * m + col] = q[col * m + row];

	}

	for (long long row = 0; row < 4; row++)
	{
		//	printf("\n��%d��", row + 1);
		for (long long col = 0; col < 4; col++){
			if (row <= 1) R(row + 1, col + 1) = -B[row * m + col];
			else R(row + 1, col + 1) = B[row * m + col];
			//	printf("%.3f\t", R(row + 1, col + 1));
		}

		//	printf("\n");
	}
	/*	for (long long row = 1; row <= R.nrows(); row++)
	{
	printf("\n��%d��", row);
	for (long long col = 1; col <= R.ncols(); col++)
	printf("%.3f\t", R(row, col));
	printf("\n");
	}*/
	hipsolverDnSorgqr_bufferSize(cusolverH, m, m, m, d_A, lda, d_W, &hwork);
	hipMalloc((void**)&h_work, sizeof(float)*hwork);
	hipsolverDnSorgqr(cusolverH, m, m, m, d_A, lda, d_W, h_work, hwork, devInfo);
	hipMemcpy(r, d_A, sizeof(float)*lda*m, hipMemcpyDeviceToHost);
	//hipMemcpy(W, d_W, sizeof(float)*m*m, hipMemcpyDeviceToHost);
	hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);


	//	float *C = (float*)malloc(lda*m * sizeof(float));


	for (long long row = 0; row < m; row++)
	{

		for (long long col = 0; col < 2; col++)
			Q(row + 1, col + 1) = -r[col * m + row];
		for (long long col = 2; col < 4; col++)
			Q(row + 1, col + 1) = r[col * m + row];
	}
	//for (long long row = 0; row < m; row++)
	//{
	//	printf("\n��%d��", row);
	//	for (long long col = 0; col < m; col++)
	//		printf("%.3f\t", Q(row + 1, col + 1));
	//	printf("\n");
	//}

	//for (int j = 0; j < m*m; j++)
	//{

	//	if (((j) % m) == 0){ printf("\n��%d��", (((j + 1) / m) + 1)); }
	//	printf("%.3f\t", C[j]);


	//}
	free(q);
	free(r);
	free(W);
	free(B);
	hipFree(&d_A);
	hipFree(&h_work);
	hipFree(&devInfo);
	hipFree(&d_work);
	hipsolverDnDestroy(cusolverH);
	HANDLE_ERROR(hipDeviceReset());
	return 0;

}