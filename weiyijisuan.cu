#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <malloc.h>
#include<time.h>
#include"newmat.h"
#include "ele.cuh"
#include "hip/hip_runtime_api.h"
#define nCpt_Max 20000

#if defined(_MSC_VER) && (_WIN64)
//#if defined(_MSC_VER) && defined(_WIN64) //correct?

#define V3DLONG long long

#else

#define V3DLONG long

#endif

typedef unsigned char UINT8_JBA;
typedef float MYFLOAT_JBA;

class DisplaceFieldF3D
{
public:
	UINT8_JBA b_transform;

	MYFLOAT_JBA sx, sy, sz; //shift of x,y,z
	DisplaceFieldF3D() { sx = sy = sz = 0; b_transform = 0; }
	DisplaceFieldF3D(double vv) { sx = sy = sz = vv; b_transform = 0; }
	void scale(double dfactor) { sx *= dfactor; sy *= dfactor; sz *= dfactor; }
	void resetToDefault() //070517
	{
		sx = 0; sy = 0; sz = 0;
		b_transform = 0;
	}
	bool copy(DisplaceFieldF3D *wp)
	{
		if (!wp) return false;
		sx = wp->sx;  sy = wp->sy;  sz = wp->sz;
		b_transform = wp->b_transform;
		return true;
	}
	bool copy(DisplaceFieldF3D &wp)
	{
		sx = wp.sx;  sy = wp.sy;  sz = wp.sz;
		b_transform = wp.b_transform;
		return true;
	}

};


__device__ void MUL(float *A, float *B, float *C, float *D, float E[4], int nCpt)//MUL(x_ori, D_x4x4_d, xmxn_K1, D_xnx4_c, x_stps, nCpt)
{
	int k, a; int p1 = 0; int p2 = 0;

	for (int i = 0; i < 4; i++)
	{
		float sum1 = 0; float sum2 = 0;
		for (k = 0; k < 4; k++)
		{
			sum1 += A[k] * B[k + p1];
		}
		for (a = 0; a < nCpt; a++)
		{
			sum2 += C[a] * D[a + p2];
		}
		E[i] = sum1 + sum2;
		p1 = p1 + 4;
		p2 = p2 + nCpt;
	}


}
__device__ void assignment(V3DLONG k, int row, int col, V3DLONG gsz1, V3DLONG gsz0, V3DLONG gfactor_x, V3DLONG gfactor_y, V3DLONG gfactor_z, float *D_RESULT_X, float *D_RESULT_Y, float *D_RESULT_Z, const float x_stps[4])
{
	//printf("\t>>gfactor_z[%d] [%d] : %f  %f  %f  %f  k=%d:\n", row, col, x_stps[0], x_stps[1], x_stps[2], x_stps[3], k);
	int id = k * 88 + row*gsz0 + col;
	float a = x_stps[1], b = x_stps[2], c = x_stps[3]; __syncthreads();
	//printf("\t>>gfactor_z[%d] [%d] : %f  %f  %f  %f  k=%d:\n", row, col, x_stps[0], a, b, c, k);
	D_RESULT_Y[id] = b - (row - 1)*gfactor_y;// printf("\t>>gfactor_z[%d] [%d] : %f  %f  %f  %f  k=%d:\n", row, col, x_stps[0], x_stps[1], x_stps[2], x_stps[3], k);

	D_RESULT_X[id] = a - (col - 1)*gfactor_x; //__syncthreads();

	D_RESULT_Z[id] = c - (k - 1)*gfactor_z;
	//	printf("\t>>gfactor_z[%d] [%d] :%d  %f  %f  %f  k=%d:\n", row, col, id,  D_RESULT_X[id], D_RESULT_Y[id], D_RESULT_Z[id], k);
}
__global__ void get_cd(int nCpt, V3DLONG k, const V3DLONG gsz1, const V3DLONG gsz0, V3DLONG gfactor_x, V3DLONG gfactor_y, V3DLONG gfactor_z,
	float * D_X, float * D_Y, float * D_Z, float *D_x4x4_d, float *D_xnx4_c, float *D_RESULT_X, float *D_RESULT_Y, float *D_RESULT_Z,
	long long x_offset, long long y_offset, long long z_offset)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= gsz1 || col >= gsz0)return;
	float x_ori[4]; float xmxn_K1[nCpt_Max]; float x_stps[4];
	x_stps[0] = 0; x_stps[1] = 0; x_stps[2] = 0; x_stps[3] = 0;
	x_ori[0] = 1.0; x_ori[1] = (col - 1)*gfactor_x + x_offset; x_ori[2] = (row - 1)*gfactor_y + y_offset; x_ori[3] = (k - 1)*gfactor_z + z_offset;
	for (int n = 0; n < nCpt; n++)
	{
		xmxn_K1[n] = -sqrt(((col - 1)*gfactor_x + x_offset - D_X[n])*((col - 1)*gfactor_x + x_offset - D_X[n]) + ((row - 1)*gfactor_y + y_offset - D_Y[n])*((row - 1)*gfactor_y + y_offset - D_Y[n]) + ((k - 1)*gfactor_z + z_offset - D_Z[n])*((k - 1)*gfactor_z + z_offset - D_Z[n]));
	}


	MUL(x_ori, D_x4x4_d, xmxn_K1, D_xnx4_c, x_stps, nCpt);


	D_RESULT_X[k*gsz1*gsz0 + row*gsz0 + col] = x_stps[1] - ((col - 1)*gfactor_x + x_offset);
	D_RESULT_Y[k*gsz1*gsz0 + row*gsz0 + col] = x_stps[2] - ((row - 1)*gfactor_y + y_offset);
	D_RESULT_Z[k*gsz1*gsz0 + row*gsz0 + col] = x_stps[3] - ((k - 1)*gfactor_z + z_offset);

}



extern "C" bool gpu_computedistance(int nCpt, const V3DLONG gsz2, const V3DLONG gsz1, const V3DLONG gsz0, V3DLONG gfactor_x, V3DLONG gfactor_y, V3DLONG gfactor_z,
	Matrix &x4x4_d, Matrix &xnx4_c, float * H_X, float * H_Y, float * H_Z, DisplaceFieldF3D *** df_local_3d, long long x_offset, long long y_offset, long long z_offset)
{
	float *D_X, *D_Y, *D_Z, *H_xmxn_K, *D_xmxn_K, *H_ori, *D_ori, *H_stps, *D_stps, *H_x4x4_d, *D_x4x4_d, *H_xnx4_c, *D_xnx4_c, *H_RESULT_X, *H_RESULT_Y, *H_RESULT_Z, *D_RESULT_X, *D_RESULT_Y, *D_RESULT_Z;
	H_xmxn_K = (float*)malloc(nCpt * sizeof(float));
	H_ori = (float*)malloc(4 * sizeof(float));
	H_stps = (float*)malloc(4 * sizeof(float));
	H_x4x4_d = (float*)malloc(4 * 4 * sizeof(float));
	H_xnx4_c = (float*)malloc(nCpt * 4 * sizeof(float));
	H_RESULT_X = (float*)malloc(gsz2 * gsz1 *gsz0* sizeof(float));
	H_RESULT_Y = (float*)malloc(gsz2 * gsz1 *gsz0* sizeof(float));
	H_RESULT_Z = (float*)malloc(gsz2 * gsz1 *gsz0* sizeof(float));

	hipMalloc((void**)&D_X, nCpt * sizeof(float));
	hipMalloc((void**)&D_Y, nCpt * sizeof(float));
	hipMalloc((void**)&D_Z, nCpt * sizeof(float));
	hipMalloc((void**)&D_xmxn_K, nCpt * sizeof(float));
	hipMalloc((void**)&D_x4x4_d, 4 * 4 * sizeof(float));
	hipMalloc((void**)&D_xnx4_c, nCpt * 4 * sizeof(float));
	hipMalloc((void**)&D_ori, 4 * sizeof(float));
	hipMalloc((void**)&D_stps, 4 * sizeof(float));
	hipMalloc((void**)&D_RESULT_X, gsz2 * gsz1 *gsz0* sizeof(float));
	hipMalloc((void**)&D_RESULT_Y, gsz2 * gsz1 *gsz0* sizeof(float));
	hipMalloc((void**)&D_RESULT_Z, gsz2 * gsz1 *gsz0* sizeof(float));
	printf("\t>>gsz2 : %d \n", gsz2);
	printf("\t>>gsz1 : %d \n", gsz1);
	printf("\t>>gsz0 : %d \n", gsz0);
	for (int i = 0; i < x4x4_d.nrows(); i++)//x4x4_d����ת��������H_X4X4_d
	{
		for (int j = 0; j < x4x4_d.ncols(); j++)
		{

			H_x4x4_d[j * x4x4_d.nrows() + i] = x4x4_d(i + 1, j + 1);

		}
	}



	for (int i = 0; i < xnx4_c.nrows(); i++)//xnx4_c����ת��������H_xnx4_c
	{
		for (int j = 0; j < xnx4_c.ncols(); j++)
		{

			H_xnx4_c[j * xnx4_c.nrows() + i] = xnx4_c(i + 1, j + 1);

		}
	}


	HANDLE_ERROR(hipMemcpy(D_X, H_X, nCpt * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_Y, H_Y, nCpt * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_Z, H_Z, nCpt * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_x4x4_d, H_x4x4_d, 4 * 4 * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_xnx4_c, H_xnx4_c, 4 * nCpt* sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_RESULT_X, H_RESULT_X, gsz2 * gsz1 *gsz0* sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_RESULT_Y, H_RESULT_Y, gsz2 * gsz1 *gsz0* sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(D_RESULT_Z, H_RESULT_Z, gsz2 * gsz1 *gsz0* sizeof(float), hipMemcpyHostToDevice));

	for (int k = 0; k < gsz2; k++)
	{
		dim3 grid((gsz0 + threads_num - 1) / threads_num, (gsz1 + threads_num - 1) / threads_num);//dim3 ����һ����ά�������������һάΪ1
		dim3 block(threads_num, threads_num);
		get_cd << <grid, block >> >(nCpt, k, gsz1, gsz0, gfactor_x, gfactor_y, gfactor_z, D_X, D_Y, D_Z, D_x4x4_d, D_xnx4_c, D_RESULT_X, D_RESULT_Y, D_RESULT_Z, x_offset, y_offset, z_offset);
		hipDeviceSynchronize();
		//printf("---------mark[%d]-------\n\n", k);
	}

	HANDLE_ERROR(hipMemcpy(H_RESULT_X, D_RESULT_X, gsz2 * gsz1 * gsz0 * sizeof(float), hipMemcpyDeviceToHost));//�豸������ ��������
	HANDLE_ERROR(hipMemcpy(H_RESULT_Y, D_RESULT_Y, gsz2 * gsz1 * gsz0 * sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(H_RESULT_Z, D_RESULT_Z, gsz2 * gsz1 * gsz0 * sizeof(float), hipMemcpyDeviceToHost));



	for (V3DLONG a = 0; a < gsz2; a++)
	{
		for (V3DLONG b = 0; b < gsz1; b++)
		{
			for (V3DLONG c = 0; c < gsz0; c++)
			{
				df_local_3d[a][b][c].sx = H_RESULT_X[a*gsz1*gsz0 + b*gsz0 + c]; //printf("\t>>gfactor_z[%d] [%d] :x=%.3f\n ", b, c, df_local_3d[a][b][c].sx);
				df_local_3d[a][b][c].sy = H_RESULT_Y[a*gsz1*gsz0 + b*gsz0 + c]; //printf("y=%.3f ", df_local_3d[a][b][c].sy);
				df_local_3d[a][b][c].sz = H_RESULT_Z[a*gsz1*gsz0 + b*gsz0 + c];// printf("z=%.3f  ", df_local_3d[a][b][c].sz);
			}
		}
	}

	free(H_xmxn_K);
	free(H_ori);
	free(H_stps);
	free(H_xnx4_c);
	free(H_x4x4_d);
	free(H_RESULT_X);
	free(H_RESULT_Y);
	free(H_RESULT_Z);

	hipFree(&D_X); hipFree(&D_ori); hipFree(&D_stps); hipFree(&D_x4x4_d); hipFree(&D_xnx4_c);
	hipFree(&D_Y); hipFree(&D_RESULT_X); hipFree(&D_RESULT_Y); hipFree(&D_RESULT_Z);
	hipFree(&D_Z); hipFree(&D_xmxn_K);
	HANDLE_ERROR(hipDeviceReset());
	return true;
}