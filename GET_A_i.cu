#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <malloc.h>
#include <iostream>
#include <chrono>
#include <ctime>
#include <time.h>
#include <stdlib.h>
#include"newmat.h"
//#include "newmat.h"



__global__ void MatrixInvert(float *A_splice, float *A_mem, int n, int row)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (i >= 2 * n || j >= n) return;

	float temp = 0;



	if (j != row)
	{
		temp = A_mem[j] / A_splice[row * 2 * n + row];
		A_splice[j * 2 * n + i] = A_splice[j * 2 * n + i] - A_splice[row * 2 * n + i] * temp;
	}
}


__global__ void Matrixend(float *A_splice, float *A_mem, int n, int row)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= 2 * n) return;

	float temp = 0;


	A_splice[row * 2 * n + i] = A_splice[row * 2 * n + i] / A_mem[row];


}


__global__ void Matrixpre(float *A_splice, int n, int row)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= 2 * n) return;

	float temp = 0;


	for (int Newrow = 0; Newrow < n; Newrow++)
	{
		if (A_splice[Newrow * 2 * n + row] != 0)
		{
			A_splice[row * 2 * n + i] = A_splice[row * 2 * n + i] + A_splice[Newrow * 2 * n + i];
			break;
		}
	}

}

__global__ void Matrixmem(float *A_mem, float *A_splice, int n, int row)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= n) return;

	A_mem[i] = A_splice[i * 2 * n + row];
}


extern "C" int gpu_A_i_new(int ncpt, const Matrix &A, Matrix &A_i)
{
	const int n = ncpt - 4;
	const int size = n * n * sizeof(float);

	float *h_A_splice = (float*)malloc(2 * size);


	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < 2 * n; j++)
		{
			if (j < n)
			{
				h_A_splice[i * 2 * n + j] = A(i + 1, j + 1);
			}
			else if (j >= n)
			{
				if (i + n == j)
				{
					h_A_splice[i * 2 * n + j] = 1;
				}
				else
				{
					h_A_splice[i * 2 * n + j] = 0;
				}
			}
		}
	}



	float *d_A_splice, *d_A_mem;
	hipMalloc((void**)&d_A_splice, 2 * size);
	hipMalloc((void**)&d_A_mem, n * sizeof(float));

	hipMemcpy(d_A_splice, h_A_splice, 2 * size, hipMemcpyHostToDevice);

	int threads = 32;
	dim3 block(threads, threads);
	dim3 grid((2 * n + threads - 1) / threads, (n + threads - 1) / threads);

	dim3 block1(threads);
	dim3 grid1((2 * n + threads - 1) / threads);


	for (int row = 0; row < n; row++)
	{
		if (h_A_splice[row * 2 * n + row] == 0)
		{
			Matrixpre << <grid1, block1 >> > (d_A_splice, n, row);
			hipDeviceSynchronize();
		}
	}


	for (int row = 0; row < n; row++)
	{
		Matrixmem << <grid1, block1 >> > (d_A_mem, d_A_splice, n, row);
		hipDeviceSynchronize();
		MatrixInvert << <grid, block >> > (d_A_splice, d_A_mem, n, row);
		hipDeviceSynchronize();
		Matrixend << < grid1, block1 >> > (d_A_splice, d_A_mem, n, row);

	}

	hipMemcpy(h_A_splice, d_A_splice, 2 * size, hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			A_i(i + 1, j + 1) = h_A_splice[i * 2 * n + n + j];
		}
	}

	free(h_A_splice);
	hipFree(d_A_splice);
	hipFree(d_A_mem);
	hipDeviceReset();

	return 0;
}